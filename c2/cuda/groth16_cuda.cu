#include "hip/hip_runtime.h"
// Copyright Supranational LLC

#include <iostream>
#include <thread>
#include <vector>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381-fp2.hpp>
#else
# error "only FEATURE_BLS12_381 is supported"
#endif

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;

typedef jacobian_t<fp2_t> point_fp2_t;
typedef xyzz_t<fp2_t> bucket_fp2_t;
typedef bucket_fp2_t::affine_t affine_fp2_t;

typedef fr_t scalar_t;

#include <msm/pippenger.cuh>
#include <msm/pippenger.hpp>

#include "groth16_ntt_h.cu"
#include "groth16_split_msm.cu"

template<class point_t, class affine_t>
static void mult(point_t& ret, const affine_t point, const scalar_t& fr,
                 size_t top = scalar_t::nbits)
{
    scalar_t::pow_t scalar;
    fr.to_scalar(scalar);

    mult(ret, point, scalar, top);
}

static thread_pool_t groth16_pool;

struct verifying_key {
    affine_t alpha_g1;
    affine_t beta_g1;
    affine_fp2_t beta_g2;
    affine_fp2_t gamma_g2;
    affine_t delta_g1;
    affine_fp2_t delta_g2;
};

struct msm_results {
    std::vector<point_t> h;
    std::vector<point_t> l;
    std::vector<point_t> a;
    std::vector<point_t> b_g1;
    std::vector<point_fp2_t> b_g2;

    msm_results(size_t num_circuits) : h(num_circuits),
                                       l(num_circuits),
                                       a(num_circuits),
                                       b_g1(num_circuits),
                                       b_g2(num_circuits) {}
};

struct groth16_proof {
    point_t::affine_t a;
    point_fp2_t::affine_t b;
    point_t::affine_t c;
};


#ifndef __CUDA_ARCH__

extern "C" {
    int blst_p1_deserialize(affine_t*, const byte[96]);
    int blst_p2_deserialize(affine_fp2_t*, const byte[192]);
}

#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>

// This class assumes that the SRS files used by filecoin have a specific file
// layout and assumes some properties of data types that are present in the file
//
// There are 3 data types in the file:
//     4-byte   big-endian unsigned integer,
//     92-byte  BLS12-381 P1 affine point,
//     192-byte BLS12-381 P2 affine point
//
// The layout of the file is as such, in order, without any padding:
//
// alpha_g1: g1 affine
// beta_g1 : g1 affine
// beta_g2 : g2 affine
// gamma_g2: g2 affine
// delta_g1: g1 affine
// delta_g2: g2 affine
// number of ic points: 4-byte big-endian unsigned integer
// ic points: g1 affines
// number of h points: 4-byte big-endian unsigned integer
// h points: g1 affines
// number of l points: 4-byte big-endian unsigned integer
// l points: g1 affines
// number of a points: 4-byte big-endian unsigned integer
// a points: g1 affines
// number of b_g1 points: 4-byte big-endian unsigned integer
// b_g1 points: g1 affines
// number of b_g2 points: 4-byte big-endian unsigned integer
// b_g2 points: g2 affines
class SRS {
private:
    // size of p1 affine and p2 affine points in the SRS file in bytes
    static const size_t p1_affine_size = 96;
    static const size_t p2_affine_size = 192;

    // 3 p1 affine and 3 p2 affine points are in the verification key. 864 bytes
    static const size_t vk_offset = p1_affine_size * 3 + p2_affine_size * 3;

    // the number of points for each of h, l, a, b_g1 and b_g2 are stored as
    // a big-endian uint32_t in the SRS file
    static const size_t four_bytes = 4;

    bool currently_initialized = false;

    struct srs_data {
        uint32_t size = 0;
        size_t off = 0; // in bytes
    };

    template<typename T>
    static T from_big_endian(const unsigned char* ptr) {
        T res = ptr[0];
        for (size_t i = 1; i < sizeof(T); i++) {
            res <<= 8;
            res |= ptr[i];
        }

        return res;
    }

    static size_t get_batch_size(uint32_t num_points, size_t num_threads) {
        size_t batch_size = (num_points + num_threads - 1) / num_threads;
        batch_size = (batch_size + 64 - 1) / 64;
        return batch_size;
    }

    void read_g1_points(affine_t* points, const byte* srs_ptr,
                        uint32_t num_points = 1)
    {
        size_t batch_size = get_batch_size(num_points, groth16_pool.size());

        const byte (*srs)[p1_affine_size] =
            reinterpret_cast<decltype(srs)>(srs_ptr);

        groth16_pool.par_map(num_points, batch_size, [&](size_t i) {
            blst_p1_deserialize(&points[i], srs[i]);
        });
    }

    void read_g2_points(affine_fp2_t* points, const byte* srs_ptr,
                        uint32_t num_points = 1)
    {
        size_t batch_size = get_batch_size(num_points, groth16_pool.size());

        const byte (*srs)[p2_affine_size] =
            reinterpret_cast<decltype(srs)>(srs_ptr);

        groth16_pool.par_map(num_points, batch_size, [&](size_t i) {
            blst_p2_deserialize(&points[i], srs[i]);
        });
    }

    srs_data data_h, data_l, data_a, data_b_g1, data_b_g2;

    SRS() {}

public:
    static SRS& get_instance() {
        static SRS instance;
        return instance;
    }

    SRS(SRS const&)            = delete;
    void operator=(SRS const&) = delete;

    verifying_key vk;
    std::vector<affine_t> h, l, a, b_g1;
    std::vector<affine_fp2_t> b_g2;

    // in case one wants to deallocate before program's end
    void reset() {
        if (!currently_initialized)
            return;

        h = std::vector<affine_t>();
        l = std::vector<affine_t>();
        a = std::vector<affine_t>();
        b_g1 = std::vector<affine_t>();
        b_g2 = std::vector<affine_fp2_t>();

        currently_initialized = false;
    }

    void read(const char* srs_path) {
        if (currently_initialized)
            reset();

        int srs_file = open(srs_path, O_RDONLY);

        struct stat st;
        fstat(srs_file, &st);
        size_t file_size = st.st_size;

        const byte* srs_ptr = (const byte*)mmap(NULL, file_size, PROT_READ,
                                                MAP_PRIVATE, srs_file, 0);
        close(srs_file);

        read_g1_points(&vk.alpha_g1, srs_ptr + 0);
        read_g1_points(&vk.beta_g1, srs_ptr + p1_affine_size);
        read_g2_points(&vk.beta_g2, srs_ptr + 2 * p1_affine_size);
        read_g2_points(&vk.gamma_g2, srs_ptr + 2 * p1_affine_size +
                                                   p2_affine_size);
        read_g1_points(&vk.delta_g1, srs_ptr + 2 * p1_affine_size +
                                               2 * p2_affine_size);
        read_g2_points(&vk.delta_g2, srs_ptr + 3 * p1_affine_size +
                                               2 * p2_affine_size);

        uint32_t vk_ic_size = from_big_endian<uint32_t>(srs_ptr + vk_offset);

        data_h.size = from_big_endian<uint32_t>(srs_ptr + vk_offset +
                                                four_bytes +
                                                vk_ic_size * p1_affine_size);
        data_h.off = vk_offset + four_bytes + vk_ic_size * p1_affine_size +
                     four_bytes;

        data_l.size = from_big_endian<uint32_t>(srs_ptr + data_h.off +
                                                data_h.size * p1_affine_size);
        data_l.off = data_h.off + data_h.size * p1_affine_size + four_bytes;

        data_a.size = from_big_endian<uint32_t>(srs_ptr + data_l.off +
                                                data_l.size * p1_affine_size);
        data_a.off = data_l.off + data_l.size * p1_affine_size + four_bytes;

        data_b_g1.size = from_big_endian<uint32_t>(srs_ptr + data_a.off +
                                                   data_a.size *
                                                   p1_affine_size);
        data_b_g1.off = data_a.off + data_a.size * p1_affine_size + four_bytes;

        data_b_g2.size = from_big_endian<uint32_t>(srs_ptr + data_b_g1.off +
                                                   data_b_g1.size *
                                                   p1_affine_size);
        data_b_g2.off = data_b_g1.off + data_b_g1.size * p1_affine_size +
                        four_bytes;

        h.resize(data_h.size);
        l.resize(data_l.size);
        a.resize(data_a.size);
        b_g1.resize(data_b_g1.size);
        b_g2.resize(data_b_g2.size);

        read_g1_points(&h[0], srs_ptr + data_h.off, data_h.size);
        read_g1_points(&l[0], srs_ptr + data_l.off, data_l.size);
        read_g1_points(&a[0], srs_ptr + data_a.off, data_a.size);
        read_g1_points(&b_g1[0], srs_ptr + data_b_g1.off, data_b_g1.size);
        read_g2_points(&b_g2[0], srs_ptr + data_b_g2.off, data_b_g2.size);

        munmap(const_cast<byte*>(srs_ptr), file_size);

        currently_initialized = true;
    }
};

extern "C"
void read_srs_c(const char* srs_path) {
    SRS::get_instance().read(srs_path);
}

extern "C"
void reset_srs_c() {
    SRS::get_instance().reset();
}

extern "C"
RustError generate_groth16_proof_c(ntt_msm_h_inputs_c& ntt_msm_h_inputs,
    msm_l_a_b_g1_b_g2_inputs_c& msm_l_a_b_g1_b_g2_inputs, size_t num_circuits,
    const fr_t r_s[], const fr_t s_s[], groth16_proof proofs[])
{
    SRS& srs = SRS::get_instance();
    verifying_key* vk = &srs.vk;

    ntt_msm_h_inputs.points_h = &srs.h[0];
    msm_l_a_b_g1_b_g2_inputs.points_l.points = &srs.l[0];
    msm_l_a_b_g1_b_g2_inputs.points_a.points = &srs.a[0];
    msm_l_a_b_g1_b_g2_inputs.points_b_g1.points = &srs.b_g1[0];
    msm_l_a_b_g1_b_g2_inputs.points_b_g2.points = &srs.b_g2[0];

    const points_c<affine_t>& points_l = msm_l_a_b_g1_b_g2_inputs.points_l;
    const points_c<affine_t>& points_a = msm_l_a_b_g1_b_g2_inputs.points_a;
    const points_c<affine_t>& points_b_g1 = msm_l_a_b_g1_b_g2_inputs.points_b_g1;
    const points_c<affine_fp2_t>& points_b_g2 = msm_l_a_b_g1_b_g2_inputs.points_b_g2;

    split_vectors split_vectors_l{num_circuits, points_l.size};
    split_vectors split_vectors_a{num_circuits, points_a.size - points_a.skip};
    split_vectors split_vectors_b{num_circuits, points_b_g1.size - points_b_g1.skip};

    std::vector<affine_t> tail_msm_l_bases,
                          tail_msm_a_bases,
                          tail_msm_b_g1_bases;
    std::vector<affine_fp2_t> tail_msm_b_g2_bases;

    msm_results results{num_circuits};

    semaphore_t barrier;
    std::atomic<bool> caught_exception{false};
    size_t n_gpus = std::min(ngpus(), num_circuits);

    std::thread prep_msm_thread([&, num_circuits]
    {
#if 1   // minimize reference passing
        const points_c<affine_t>& points_l = msm_l_a_b_g1_b_g2_inputs.points_l;
        const points_c<affine_t>& points_a = msm_l_a_b_g1_b_g2_inputs.points_a;
        const points_c<affine_t>& points_b_g1 = msm_l_a_b_g1_b_g2_inputs.points_b_g1;
        const points_c<affine_fp2_t>& points_b_g2 = msm_l_a_b_g1_b_g2_inputs.points_b_g2;
#endif
        const fr_t** input_assignments = msm_l_a_b_g1_b_g2_inputs.input_assignments;
        const fr_t** aux_assignments = msm_l_a_b_g1_b_g2_inputs.aux_assignments;

        size_t input_assignment_size = msm_l_a_b_g1_b_g2_inputs.input_assignment_size;
        size_t aux_assignment_size = msm_l_a_b_g1_b_g2_inputs.aux_assignment_size;

        // pre-processing step
        const fr_t* input_assignment0 = input_assignments[0];
        const fr_t* aux_assignment0 = aux_assignments[0];

        size_t l_counter = 0,
               a_counter = points_a.skip,
               b_counter = points_b_g1.skip;

        for (size_t i = 0; i < aux_assignment_size; i += chunk_bits) {
            uint64_t a_chunk = points_a.density_map[i / chunk_bits];
            uint64_t b_chunk = points_b_g1.density_map[i / chunk_bits];

            for (size_t j = 0; j < chunk_bits; j++) {
                if (i + j >= aux_assignment_size) break;

                const fr_t& scalar = aux_assignment0[i + j];

                bool a_dense = a_chunk & 1;
                bool b_g1_dense = b_chunk & 1;

                if (!scalar.is_zero() && !scalar.is_one()) {
                    l_counter++;
                    if (a_dense)
                        a_counter++;
                    if (b_g1_dense)
                        b_counter++;
                }

                a_chunk >>= 1;
                b_chunk >>= 1;
            }
        }
        // end of pre-processing step

        if (caught_exception)
            return;

        split_vectors_l.tail_msms_resize(l_counter);
        split_vectors_a.tail_msms_resize(a_counter);
        split_vectors_b.tail_msms_resize(b_counter);

        tail_msm_l_bases.resize(l_counter);
        tail_msm_a_bases.resize(a_counter);
        tail_msm_b_g1_bases.resize(b_counter);
        tail_msm_b_g2_bases.resize(b_counter);

        groth16_pool.par_map(num_circuits, [&](size_t c) {
            uint64_t bit_vector_a_chunk = 0, bit_vector_b_chunk = 0;
            size_t a_chunk_counter = 0, b_chunk_counter = 0;
            size_t a_chunk_cursor = 0, b_chunk_cursor = 0;

            uint32_t points_a_cursor = 0, points_b_cursor = 0;
            size_t l_meaningful_scalars_counter = 0;
            size_t a_meaningful_scalars_counter = 0;
            size_t b_meaningful_scalars_counter = 0;

            for (size_t i = 0; i < input_assignment_size; i++) {
                const fr_t& scalar = input_assignments[c][i];

                if (i < points_a.skip) {
                    if (c == 0)
                        split_vectors_a.tail_msm_indices[a_meaningful_scalars_counter] = points_a_cursor;
                    split_vectors_a.tail_msm_scalars[c][a_meaningful_scalars_counter] = scalar;

                    a_meaningful_scalars_counter++;
                    points_a_cursor++;
                }

                if (i < points_b_g1.skip) {
                    if (c == 0)
                        split_vectors_b.tail_msm_indices[b_meaningful_scalars_counter] = points_b_cursor;
                    split_vectors_b.tail_msm_scalars[c][b_meaningful_scalars_counter] = scalar;

                    b_meaningful_scalars_counter++;
                    points_b_cursor++;
                }
            }

            if (caught_exception)
                return;

            for (size_t i = 0; i < aux_assignment_size; i += chunk_bits) {

                uint64_t a_chunk = points_a.density_map[i / chunk_bits];
                uint64_t b_chunk = points_b_g1.density_map[i / chunk_bits];

                uint64_t bit_vector_l_chunk = 0;

                for (size_t j = 0; j < chunk_bits; j++) {
                    if (i + j >= aux_assignment_size) break;

                    const fr_t& scalar = aux_assignments[c][i + j];

                    bool a_dense = a_chunk & 1;
                    bool b_g1_dense = b_chunk & 1;

                    if (scalar.is_one()) {
                        bit_vector_l_chunk |= ((uint64_t)1 << j);
                    }
                    else if (!scalar.is_zero()) {
                        if (c == 0)
                            split_vectors_l.tail_msm_indices[l_meaningful_scalars_counter] = (uint32_t)(i + j);
                        split_vectors_l.tail_msm_scalars[c][l_meaningful_scalars_counter] = scalar;

                        l_meaningful_scalars_counter++;
                    }

                    if (a_dense) {
                        if (scalar.is_one()) {
                            bit_vector_a_chunk |= ((uint64_t)1 << a_chunk_counter);
                        }
                        else if (!scalar.is_zero()) {
                            if (c == 0)
                                split_vectors_a.tail_msm_indices[a_meaningful_scalars_counter] = points_a_cursor;
                            split_vectors_a.tail_msm_scalars[c][a_meaningful_scalars_counter] = scalar;

                            a_meaningful_scalars_counter++;
                        }

                        a_chunk_counter++;
                        points_a_cursor++;
                    }

                    if (b_g1_dense) {
                        if (scalar.is_one()) {
                            bit_vector_b_chunk |= ((uint64_t)1 << b_chunk_counter);
                        }
                        else if (!scalar.is_zero()) {
                            if (c == 0)
                                split_vectors_b.tail_msm_indices[b_meaningful_scalars_counter] = points_b_cursor;
                            split_vectors_b.tail_msm_scalars[c][b_meaningful_scalars_counter] = scalar;

                            b_meaningful_scalars_counter++;
                        }

                        b_chunk_counter++;
                        points_b_cursor++;
                    }

                    if (a_chunk_counter == chunk_bits) {
                        split_vectors_a.bit_vector[c][a_chunk_cursor] = bit_vector_a_chunk;
                        a_chunk_counter = 0;
                        bit_vector_a_chunk = 0;
                        a_chunk_cursor++;
                    }

                    if (b_chunk_counter == chunk_bits) {
                        split_vectors_b.bit_vector[c][b_chunk_cursor] = bit_vector_b_chunk;
                        b_chunk_counter = 0;
                        bit_vector_b_chunk = 0;
                        b_chunk_cursor++;
                    }

                    a_chunk >>= 1;
                    b_chunk >>= 1;
                }

                split_vectors_l.bit_vector[c][i / chunk_bits] = bit_vector_l_chunk;
            }
        });

        if (caught_exception)
            return;

        for (size_t i = 0; i < l_counter; i++)
            tail_msm_l_bases[i] = points_l[split_vectors_l.tail_msm_indices[i]];

        for (size_t i = 0; i < a_counter; i++)
            tail_msm_a_bases[i] = points_a[split_vectors_a.tail_msm_indices[i]];

        for (size_t i = 0; i < b_counter; i++) {
            tail_msm_b_g1_bases[i] = points_b_g1[split_vectors_b.tail_msm_indices[i]];
            tail_msm_b_g2_bases[i] = points_b_g2[split_vectors_b.tail_msm_indices[i]];
        }

        for (size_t i = 0; i < n_gpus; i++)
            barrier.notify();

        if (caught_exception)
            return;

        // tail MSM b_g2 - on CPU
        for (size_t c = 0; c < num_circuits; c++) {
            mult_pippenger<bucket_fp2_t>(results.b_g2[c],
                tail_msm_b_g2_bases, split_vectors_b.tail_msm_scalars[c],
                true, &groth16_pool);

            if (caught_exception)
                return;
        }
    });

    batch_add_results batch_add_res{num_circuits};
    std::vector<std::thread> per_gpu;
    RustError ret{hipSuccess};

    for (size_t tid = 0; tid < n_gpus; tid++) {
        per_gpu.emplace_back(std::thread([&, tid, n_gpus](size_t num_circuits)
        {
#if 1   // minimize reference passing
            const points_c<affine_t>& points_l = msm_l_a_b_g1_b_g2_inputs.points_l;
            const points_c<affine_t>& points_a = msm_l_a_b_g1_b_g2_inputs.points_a;
            const points_c<affine_t>& points_b_g1 = msm_l_a_b_g1_b_g2_inputs.points_b_g1;
            const points_c<affine_fp2_t>& points_b_g2 = msm_l_a_b_g1_b_g2_inputs.points_b_g2;
#endif
            const gpu_t& gpu = select_gpu(tid);

            size_t rem = num_circuits % n_gpus;
            num_circuits /= n_gpus;
            num_circuits += tid < rem;
            size_t circuit0 = tid * num_circuits;
            if (tid >= rem)
                circuit0 += rem;

            try {
                {
                    size_t d_a_sz = sizeof(fr_t) << ntt_msm_h_inputs.lg_domain_size;
                    gpu_ptr_t<fr_t> d_a{(scalar_t*)gpu.Dmalloc(d_a_sz)};

                    for (size_t c = 0; c < num_circuits; c++) {
                        ntt_msm_h::execute_ntt_msm_h(gpu, d_a, ntt_msm_h_inputs,
                                                     circuit0 + c, &results.h[0]);
                        if (caught_exception)
                            return;
                    }
                }

                barrier.wait();

                if (caught_exception)
                    return;

                // batch addition L - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 points_l, split_vectors_l,
                                                 &batch_add_res.l[circuit0]);
                if (caught_exception)
                    return;

                // batch addition a - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 points_a, split_vectors_a,
                                                 &batch_add_res.a[circuit0]);
                if (caught_exception)
                    return;

                // batch addition b_g1 - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 points_b_g1, split_vectors_b,
                                                 &batch_add_res.b_g1[circuit0]);
                if (caught_exception)
                    return;

                // batch addition b_g2 - on GPU
                execute_batch_addition<bucket_fp2_t>(gpu, circuit0, num_circuits,
                                                     points_b_g2, split_vectors_b,
                                                     &batch_add_res.b_g2[circuit0]);
                if (caught_exception)
                    return;

                {
                    msm_t<bucket_t, point_t, affine_t, scalar_t> msm{nullptr,
                        tail_msm_l_bases.size()};

                    for (size_t c = circuit0; c < circuit0+num_circuits; c++) {
                        // tail MSM l - on GPU
                        msm.invoke(results.l[c], tail_msm_l_bases,
                                   split_vectors_l.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;

                        // tail MSM a - on GPU
                        msm.invoke(results.a[c], tail_msm_a_bases,
                                   split_vectors_a.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;

                        // tail MSM b_g1 - on GPU
                        msm.invoke(results.b_g1[c], tail_msm_b_g1_bases,
                                   split_vectors_b.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;
                    }
                }
            } catch (const cuda_error& e) {
                bool already = caught_exception.exchange(true);
                if (!already) {
                    for (size_t i = 1; i < n_gpus; i++)
                        barrier.notify();
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
                    ret = RustError{e.code(), e.what()};
#else
                    ret = RustError{e.code()};
#endif
                }
                gpu.sync();
            }
        }, num_circuits));
    }

    prep_msm_thread.join();
    for (auto& tid : per_gpu)
        tid.join();

    if (caught_exception)
        return ret;

    for (size_t circuit = 0; circuit < num_circuits; circuit++) {
        results.l[circuit].add(batch_add_res.l[circuit]);
        results.a[circuit].add(batch_add_res.a[circuit]);
        results.b_g1[circuit].add(batch_add_res.b_g1[circuit]);
        results.b_g2[circuit].add(batch_add_res.b_g2[circuit]);

        fr_t r = r_s[circuit], s = s_s[circuit];
        fr_t rs = r * s;
        // we want the scalars to be in Montomery form when passing them to
        // "mult" routine

        point_t g_a, g_c, a_answer, b1_answer, vk_delta_g1_rs, vk_alpha_g1_s,
                vk_beta_g1_r;
        point_fp2_t g_b;

        mult(vk_delta_g1_rs, vk->delta_g1, rs);
        mult(vk_alpha_g1_s, vk->alpha_g1, s);
        mult(vk_beta_g1_r, vk->beta_g1, r);

        mult(b1_answer, results.b_g1[circuit], r);

        // A
        mult(g_a, vk->delta_g1, r);
        g_a.add(vk->alpha_g1);
        g_a.add(results.a[circuit]);

        // B
        mult(g_b, vk->delta_g2, s);
        g_b.add(vk->beta_g2);
        g_b.add(results.b_g2[circuit]);

        // C
        mult(g_c, results.a[circuit], s);
        g_c.add(b1_answer);
        g_c.add(vk_delta_g1_rs);
        g_c.add(vk_alpha_g1_s);
        g_c.add(vk_beta_g1_r);
        g_c.add(results.h[circuit]);
        g_c.add(results.l[circuit]);

        // to affine
        proofs[circuit].a = g_a;
        proofs[circuit].b = g_b;
        proofs[circuit].c = g_c;
    }

    return ret;
}

#endif
