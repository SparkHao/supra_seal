#include "hip/hip_runtime.h"
// Copyright Supranational LLC

#include <iostream>
#include <thread>
#include <vector>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381-fp2.hpp>
#else
# error "only FEATURE_BLS12_381 is supported"
#endif

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;

typedef jacobian_t<fp2_t> point_fp2_t;
typedef xyzz_t<fp2_t> bucket_fp2_t;
typedef bucket_fp2_t::affine_t affine_fp2_t;

typedef fr_t scalar_t;

#include <msm/pippenger.cuh>
#include <msm/pippenger.hpp>

#include "groth16_ntt_h.cu"
#include "groth16_split_msm.cu"

template<class point_t, class affine_t>
static void mult(point_t& ret, const affine_t point, const scalar_t& fr,
                 size_t top = scalar_t::nbits)
{
    scalar_t::pow_t scalar;
    fr.to_scalar(scalar);

    mult(ret, point, scalar, top);
}

static thread_pool_t groth16_pool;

struct msm_results {
    std::vector<point_t> h;
    std::vector<point_t> l;
    std::vector<point_t> a;
    std::vector<point_t> b_g1;
    std::vector<point_fp2_t> b_g2;

    msm_results(size_t num_circuits) : h(num_circuits),
                                       l(num_circuits),
                                       a(num_circuits),
                                       b_g1(num_circuits),
                                       b_g2(num_circuits) {}
};

struct groth16_proof {
    point_t::affine_t a;
    point_fp2_t::affine_t b;
    point_t::affine_t c;
};

#ifndef __CUDA_ARCH__

#include "groth16_srs.cuh"

extern "C"
RustError generate_groth16_proof_c(ntt_msm_h_inputs_c& ntt_msm_h_inputs,
    msm_l_a_b_g1_b_g2_inputs_c& msm_l_a_b_g1_b_g2_inputs, size_t num_circuits,
    const fr_t r_s[], const fr_t s_s[], groth16_proof proofs[], SRS& srs)
{
    const verifying_key* vk = &srs.get_vk();

    ntt_msm_h_inputs.points_h = srs.get_h().data();
    msm_l_a_b_g1_b_g2_inputs.points_l.points = srs.get_l().data();
    msm_l_a_b_g1_b_g2_inputs.points_a.points = srs.get_a().data();
    msm_l_a_b_g1_b_g2_inputs.points_b_g1.points = srs.get_b_g1().data();
    msm_l_a_b_g1_b_g2_inputs.points_b_g2.points = srs.get_b_g2().data();

    const points_c<affine_t>& points_l = msm_l_a_b_g1_b_g2_inputs.points_l;
    const points_c<affine_t>& points_a = msm_l_a_b_g1_b_g2_inputs.points_a;
    const points_c<affine_t>& points_b_g1 = msm_l_a_b_g1_b_g2_inputs.points_b_g1;
    const points_c<affine_fp2_t>& points_b_g2 = msm_l_a_b_g1_b_g2_inputs.points_b_g2;

    split_vectors split_vectors_l{num_circuits, points_l.size};
    split_vectors split_vectors_a{num_circuits, points_a.size - points_a.skip};
    split_vectors split_vectors_b{num_circuits, points_b_g1.size - points_b_g1.skip};

    std::vector<affine_t> tail_msm_l_bases,
                          tail_msm_a_bases,
                          tail_msm_b_g1_bases;
    std::vector<affine_fp2_t> tail_msm_b_g2_bases;

    msm_results results{num_circuits};

    semaphore_t barrier;
    std::atomic<bool> caught_exception{false};
    size_t n_gpus = std::min(ngpus(), num_circuits);

    std::thread prep_msm_thread([&, num_circuits]
    {
#if 1   // minimize reference passing
        const points_c<affine_t>& points_l = msm_l_a_b_g1_b_g2_inputs.points_l;
        const points_c<affine_t>& points_a = msm_l_a_b_g1_b_g2_inputs.points_a;
        const points_c<affine_t>& points_b_g1 = msm_l_a_b_g1_b_g2_inputs.points_b_g1;
        const points_c<affine_fp2_t>& points_b_g2 = msm_l_a_b_g1_b_g2_inputs.points_b_g2;
#endif
        const fr_t** input_assignments = msm_l_a_b_g1_b_g2_inputs.input_assignments;
        const fr_t** aux_assignments = msm_l_a_b_g1_b_g2_inputs.aux_assignments;

        size_t input_assignment_size = msm_l_a_b_g1_b_g2_inputs.input_assignment_size;
        size_t aux_assignment_size = msm_l_a_b_g1_b_g2_inputs.aux_assignment_size;

        // pre-processing step
        const fr_t* input_assignment0 = input_assignments[0];
        const fr_t* aux_assignment0 = aux_assignments[0];

        size_t l_counter = 0,
               a_counter = points_a.skip,
               b_counter = points_b_g1.skip;

        for (size_t i = 0; i < aux_assignment_size; i += chunk_bits) {
            uint64_t a_chunk = points_a.density_map[i / chunk_bits];
            uint64_t b_chunk = points_b_g1.density_map[i / chunk_bits];

            for (size_t j = 0; j < chunk_bits; j++) {
                if (i + j >= aux_assignment_size) break;

                const fr_t& scalar = aux_assignment0[i + j];

                bool a_dense = a_chunk & 1;
                bool b_g1_dense = b_chunk & 1;

                if (!scalar.is_zero() && !scalar.is_one()) {
                    l_counter++;
                    if (a_dense)
                        a_counter++;
                    if (b_g1_dense)
                        b_counter++;
                }

                a_chunk >>= 1;
                b_chunk >>= 1;
            }
        }
        // end of pre-processing step

        if (caught_exception)
            return;

        split_vectors_l.tail_msms_resize(l_counter);
        split_vectors_a.tail_msms_resize(a_counter);
        split_vectors_b.tail_msms_resize(b_counter);

        tail_msm_l_bases.resize(l_counter);
        tail_msm_a_bases.resize(a_counter);
        tail_msm_b_g1_bases.resize(b_counter);
        tail_msm_b_g2_bases.resize(b_counter);

        groth16_pool.par_map(num_circuits, [&](size_t c) {
            uint64_t bit_vector_a_chunk = 0, bit_vector_b_chunk = 0;
            size_t a_chunk_counter = 0, b_chunk_counter = 0;
            size_t a_chunk_cursor = 0, b_chunk_cursor = 0;

            uint32_t points_a_cursor = 0, points_b_cursor = 0;
            size_t l_meaningful_scalars_counter = 0;
            size_t a_meaningful_scalars_counter = 0;
            size_t b_meaningful_scalars_counter = 0;

            for (size_t i = 0; i < input_assignment_size; i++) {
                const fr_t& scalar = input_assignments[c][i];

                if (i < points_a.skip) {
                    if (c == 0)
                        split_vectors_a.tail_msm_indices[a_meaningful_scalars_counter] = points_a_cursor;
                    split_vectors_a.tail_msm_scalars[c][a_meaningful_scalars_counter] = scalar;

                    a_meaningful_scalars_counter++;
                    points_a_cursor++;
                }

                if (i < points_b_g1.skip) {
                    if (c == 0)
                        split_vectors_b.tail_msm_indices[b_meaningful_scalars_counter] = points_b_cursor;
                    split_vectors_b.tail_msm_scalars[c][b_meaningful_scalars_counter] = scalar;

                    b_meaningful_scalars_counter++;
                    points_b_cursor++;
                }
            }

            if (caught_exception)
                return;

            for (size_t i = 0; i < aux_assignment_size; i += chunk_bits) {

                uint64_t a_chunk = points_a.density_map[i / chunk_bits];
                uint64_t b_chunk = points_b_g1.density_map[i / chunk_bits];

                uint64_t bit_vector_l_chunk = 0;

                for (size_t j = 0; j < chunk_bits; j++) {
                    if (i + j >= aux_assignment_size) break;

                    const fr_t& scalar = aux_assignments[c][i + j];

                    bool a_dense = a_chunk & 1;
                    bool b_g1_dense = b_chunk & 1;

                    if (scalar.is_one()) {
                        bit_vector_l_chunk |= ((uint64_t)1 << j);
                    }
                    else if (!scalar.is_zero()) {
                        if (c == 0)
                            split_vectors_l.tail_msm_indices[l_meaningful_scalars_counter] = (uint32_t)(i + j);
                        split_vectors_l.tail_msm_scalars[c][l_meaningful_scalars_counter] = scalar;

                        l_meaningful_scalars_counter++;
                    }

                    if (a_dense) {
                        if (scalar.is_one()) {
                            bit_vector_a_chunk |= ((uint64_t)1 << a_chunk_counter);
                        }
                        else if (!scalar.is_zero()) {
                            if (c == 0)
                                split_vectors_a.tail_msm_indices[a_meaningful_scalars_counter] = points_a_cursor;
                            split_vectors_a.tail_msm_scalars[c][a_meaningful_scalars_counter] = scalar;

                            a_meaningful_scalars_counter++;
                        }

                        a_chunk_counter++;
                        points_a_cursor++;
                    }

                    if (b_g1_dense) {
                        if (scalar.is_one()) {
                            bit_vector_b_chunk |= ((uint64_t)1 << b_chunk_counter);
                        }
                        else if (!scalar.is_zero()) {
                            if (c == 0)
                                split_vectors_b.tail_msm_indices[b_meaningful_scalars_counter] = points_b_cursor;
                            split_vectors_b.tail_msm_scalars[c][b_meaningful_scalars_counter] = scalar;

                            b_meaningful_scalars_counter++;
                        }

                        b_chunk_counter++;
                        points_b_cursor++;
                    }

                    if (a_chunk_counter == chunk_bits) {
                        split_vectors_a.bit_vector[c][a_chunk_cursor] = bit_vector_a_chunk;
                        a_chunk_counter = 0;
                        bit_vector_a_chunk = 0;
                        a_chunk_cursor++;
                    }

                    if (b_chunk_counter == chunk_bits) {
                        split_vectors_b.bit_vector[c][b_chunk_cursor] = bit_vector_b_chunk;
                        b_chunk_counter = 0;
                        bit_vector_b_chunk = 0;
                        b_chunk_cursor++;
                    }

                    a_chunk >>= 1;
                    b_chunk >>= 1;
                }

                split_vectors_l.bit_vector[c][i / chunk_bits] = bit_vector_l_chunk;
            }
        });

        if (caught_exception)
            return;

        for (size_t i = 0; i < l_counter; i++)
            tail_msm_l_bases[i] = points_l[split_vectors_l.tail_msm_indices[i]];

        for (size_t i = 0; i < a_counter; i++)
            tail_msm_a_bases[i] = points_a[split_vectors_a.tail_msm_indices[i]];

        for (size_t i = 0; i < b_counter; i++) {
            tail_msm_b_g1_bases[i] = points_b_g1[split_vectors_b.tail_msm_indices[i]];
            tail_msm_b_g2_bases[i] = points_b_g2[split_vectors_b.tail_msm_indices[i]];
        }

        for (size_t i = 0; i < n_gpus; i++)
            barrier.notify();

        if (caught_exception)
            return;

        // tail MSM b_g2 - on CPU
        for (size_t c = 0; c < num_circuits; c++) {
            mult_pippenger<bucket_fp2_t>(results.b_g2[c],
                tail_msm_b_g2_bases, split_vectors_b.tail_msm_scalars[c],
                true, &groth16_pool);

            if (caught_exception)
                return;
        }
    });

    batch_add_results batch_add_res{num_circuits};
    std::vector<std::thread> per_gpu;
    RustError ret{hipSuccess};

    for (size_t tid = 0; tid < n_gpus; tid++) {
        per_gpu.emplace_back(std::thread([&, tid, n_gpus](size_t num_circuits)
        {
#if 1   // minimize reference passing
            const points_c<affine_t>& points_l = msm_l_a_b_g1_b_g2_inputs.points_l;
            const points_c<affine_t>& points_a = msm_l_a_b_g1_b_g2_inputs.points_a;
            const points_c<affine_t>& points_b_g1 = msm_l_a_b_g1_b_g2_inputs.points_b_g1;
            const points_c<affine_fp2_t>& points_b_g2 = msm_l_a_b_g1_b_g2_inputs.points_b_g2;
#endif
            const gpu_t& gpu = select_gpu(tid);

            size_t rem = num_circuits % n_gpus;
            num_circuits /= n_gpus;
            num_circuits += tid < rem;
            size_t circuit0 = tid * num_circuits;
            if (tid >= rem)
                circuit0 += rem;

            try {
                {
                    size_t d_a_sz = sizeof(fr_t) << ntt_msm_h_inputs.lg_domain_size;
                    gpu_ptr_t<fr_t> d_a{(scalar_t*)gpu.Dmalloc(d_a_sz)};

                    for (size_t c = 0; c < num_circuits; c++) {
                        ntt_msm_h::execute_ntt_msm_h(gpu, d_a, ntt_msm_h_inputs,
                                                     circuit0 + c, &results.h[0]);
                        if (caught_exception)
                            return;
                    }
                }

                barrier.wait();

                if (caught_exception)
                    return;

                // batch addition L - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 points_l, split_vectors_l,
                                                 &batch_add_res.l[circuit0]);
                if (caught_exception)
                    return;

                // batch addition a - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 points_a, split_vectors_a,
                                                 &batch_add_res.a[circuit0]);
                if (caught_exception)
                    return;

                // batch addition b_g1 - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 points_b_g1, split_vectors_b,
                                                 &batch_add_res.b_g1[circuit0]);
                if (caught_exception)
                    return;

                // batch addition b_g2 - on GPU
                execute_batch_addition<bucket_fp2_t>(gpu, circuit0, num_circuits,
                                                     points_b_g2, split_vectors_b,
                                                     &batch_add_res.b_g2[circuit0]);
                if (caught_exception)
                    return;

                {
                    msm_t<bucket_t, point_t, affine_t, scalar_t> msm{nullptr,
                        tail_msm_l_bases.size()};

                    for (size_t c = circuit0; c < circuit0+num_circuits; c++) {
                        // tail MSM l - on GPU
                        msm.invoke(results.l[c], tail_msm_l_bases,
                                   split_vectors_l.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;

                        // tail MSM a - on GPU
                        msm.invoke(results.a[c], tail_msm_a_bases,
                                   split_vectors_a.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;

                        // tail MSM b_g1 - on GPU
                        msm.invoke(results.b_g1[c], tail_msm_b_g1_bases,
                                   split_vectors_b.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;
                    }
                }
            } catch (const cuda_error& e) {
                bool already = caught_exception.exchange(true);
                if (!already) {
                    for (size_t i = 1; i < n_gpus; i++)
                        barrier.notify();
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
                    ret = RustError{e.code(), e.what()};
#else
                    ret = RustError{e.code()};
#endif
                }
                gpu.sync();
            }
        }, num_circuits));
    }

    prep_msm_thread.join();
    for (auto& tid : per_gpu)
        tid.join();

    if (caught_exception)
        return ret;

    for (size_t circuit = 0; circuit < num_circuits; circuit++) {
        results.l[circuit].add(batch_add_res.l[circuit]);
        results.a[circuit].add(batch_add_res.a[circuit]);
        results.b_g1[circuit].add(batch_add_res.b_g1[circuit]);
        results.b_g2[circuit].add(batch_add_res.b_g2[circuit]);

        fr_t r = r_s[circuit], s = s_s[circuit];
        fr_t rs = r * s;
        // we want the scalars to be in Montomery form when passing them to
        // "mult" routine

        point_t g_a, g_c, a_answer, b1_answer, vk_delta_g1_rs, vk_alpha_g1_s,
                vk_beta_g1_r;
        point_fp2_t g_b;

        mult(vk_delta_g1_rs, vk->delta_g1, rs);
        mult(vk_alpha_g1_s, vk->alpha_g1, s);
        mult(vk_beta_g1_r, vk->beta_g1, r);

        mult(b1_answer, results.b_g1[circuit], r);

        // A
        mult(g_a, vk->delta_g1, r);
        g_a.add(vk->alpha_g1);
        g_a.add(results.a[circuit]);

        // B
        mult(g_b, vk->delta_g2, s);
        g_b.add(vk->beta_g2);
        g_b.add(results.b_g2[circuit]);

        // C
        mult(g_c, results.a[circuit], s);
        g_c.add(b1_answer);
        g_c.add(vk_delta_g1_rs);
        g_c.add(vk_alpha_g1_s);
        g_c.add(vk_beta_g1_r);
        g_c.add(results.h[circuit]);
        g_c.add(results.l[circuit]);

        // to affine
        proofs[circuit].a = g_a;
        proofs[circuit].b = g_b;
        proofs[circuit].c = g_c;
    }

    return ret;
}

#endif
