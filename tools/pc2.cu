#include "hip/hip_runtime.h"
// g++ -g -Wall -Wextra -Werror -Wno-subobject-linkage -march=native -O3 src/tools/tree_r.cpp -Isrc/poseidon -Ideps/sppark -Ideps/blst/src -L deps/blst -lblst

// Only supports constant arity 8 throughout the tree (2KB, 32G, etc);
//
// arguments
//  last_layer_filename
// optional arguments
//  data_filename - This indicates whether or not we have a CC sector

#include <cstdint>             // uint*
#include <sys/mman.h>          // mapping
#include <sys/stat.h>          // file stats
#include <cassert>             // assertions
#include <cmath>               // log2
#include <fcntl.h>             // file open
#include <unistd.h>            // file close
#include <iostream>            // printing
#include <iomanip>             // printing
#include <chrono>              // time

#include "../pc2/cuda/pc2.cu"

#ifndef __CUDA_ARCH__
#include "../pc1/tree_r.hpp"
#include "../util/debug_helpers.cpp"
#include "../sealing/sector_parameters.cpp"

void usage(char* argv[]) {
  std::cout << "If no staged data file, CC is assumed" << std::endl;
  std::cout << "Usage: " << argv[0] << " [OPTIONS]" << std::endl;
  std::cout << "-h        Print help message" << std::endl;
  std::cout << "-d <path> Staged data file" << std::endl;
  std::cout << "-i <path> Input cache directory" << std::endl;
  std::cout << "-o <path> Output directory" << std::endl;
  exit(0);
}

void gpu_single_pc2(std::string config_filename,
                    std::string cache_dir,
                    std::string data_filename,
                    std::string output_dir) {
  topology_t topology(config_filename.c_str());
  set_core_affinity(topology.pc2_hasher);

  // Get the sector size
  size_t sector_size = SECTOR_SIZE;
  SectorParameters params(sector_size);

  // Construct the layer filenames
  std::vector<std::string> layer_filenames;
  const size_t MAX = 256;
  char fname[MAX];
  const char* layer_filename_template = "%s/sc-02-data-layer-%d.dat";
  for (size_t i = 0; i < params.GetNumLayers(); i++) {
    snprintf(fname, MAX, layer_filename_template, cache_dir.c_str(), i + 1);
    layer_filenames.push_back(fname);
  }
  
  // Total number of streams across all GPUs
  size_t stream_count = 64;

  // Batch size in nodes. Each node includes all parallel sectors
  size_t batch_size = 64 * 64 * 8;
  
  // Nodes to read per partition
  size_t nodes_to_read = params.GetNumNodes() / params.GetNumTreeRCFiles();

  streaming_node_reader_t<sealing_config1_t> node_reader
    (params, layer_filenames, sector_size);
  
  // Allocate storage for 2x the streams to support tree-c and tree-r
  node_reader.alloc_slots(stream_count * 2, params.GetNumLayers() * batch_size, true);

  bool tree_r_only = false;
  const char* data_filenames[1];
  if (!data_filename.empty()) {
    data_filenames[0] = data_filename.c_str();
  } else {
    data_filenames[0] = nullptr;
  }
  pc2_hash<sealing_config1_t>(params, topology, tree_r_only, node_reader,
                              nodes_to_read, batch_size, stream_count,
                              data_filenames, output_dir.c_str());
}

int main(int argc, char* argv[]) {
  int  opt   = 0;
  std::string data_filename       = "";
  std::string cache_dir           = "";
  std::string out_dir             = ".";
  std::string config_filename     = "demos/rust/supra_seal.cfg";
  
  while ((opt = getopt(argc, argv, "c:i:d:o:h")) != -1) {
    switch(opt) {
      case 'c':
        std::cout << "config file               " << optarg << std::endl;
        config_filename = optarg;
        break;
      case 'i':
        std::cout << "input cache_dir           " << optarg << std::endl;
        cache_dir = optarg;
        break;
      case 'd':
        std::cout << "data_filename input       " << optarg << std::endl;
        data_filename = optarg;
        break;
      case 'o':
        std::cout << "out_dir                   " << optarg << std::endl;
        out_dir = optarg;
        break;
      case 'h':
      case ':':
      case '?':
        usage(argv);
        break;
    }
  }

  if (cache_dir.empty()) {
    printf("-c <cache_dir> must be specified\n");
    usage(argv);
  }

  gpu_single_pc2(config_filename, cache_dir, data_filename, out_dir);
  return 0;
}
#endif
