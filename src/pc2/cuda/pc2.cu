// Copyright Supranational LLC

#include "../../poseidon/cuda/poseidon.cu"
#include "../../util/debug_helpers.hpp"
#include "host_ptr_t.hpp"

#ifndef __CUDA_ARCH__

#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/mman.h>

#include <filesystem>
#include <chrono>
#include "../planner.cpp"
#include "pc2.cuh"
#include "cuda_lambda_t.hpp"
#include "../../util/util.hpp"

template<class C>
pc2_t<C>::pc2_t(SectorParameters& _params, column_reader_t<C>& _reader,
                size_t _nodes_to_read, size_t _batch_size,
                size_t _stream_count, int _write_core, const char* _output_dir) :
  params(_params),
  reader(_reader),
  nodes_to_read(_nodes_to_read),
  batch_size(_batch_size),
  tree_c_address(params.GetNumNodes() / params.GetNumTreeRCFiles(),
                 TREE_ARITY, NODE_SIZE, 0),
  tree_r_address(params.GetNumNodes() / params.GetNumTreeRCFiles(),
                 TREE_ARITY, NODE_SIZE, params.GetNumTreeRDiscardRows() + 1),
  stream_count(_stream_count),
  tree_c_partition_roots(C::PARALLEL_SECTORS * TREE_ARITY),
  tree_r_partition_roots(C::PARALLEL_SECTORS * TREE_ARITY),
  gpu_results_c(_batch_size * C::PARALLEL_SECTORS / TREE_ARITY * stream_count),
  gpu_results_r(_batch_size * C::PARALLEL_SECTORS / TREE_ARITY * stream_count),
  host_buf_storage(num_host_bufs * batch_size * C::PARALLEL_SECTORS),
  write_core(_write_core),
  output_dir(_output_dir)
{
  assert (TREE_ARITY == params.GetNumTreeRCArity());
  assert (nodes_to_read % stream_count == 0);

  p_aux_template = "%s/p_aux-s-%03ld.dat";
  
  // Open all tree-c and tree-r files
  const char* tree_c_filename_template = "%s/sc-02-data-tree-c-s-%03ld-%ld.dat";
  const char* tree_r_filename_template = "%s/sc-02-data-tree-r-last-s-%03ld-%ld.dat";
  std::filesystem::create_directory(output_dir);
  for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
    tree_c_fds[i].resize(params.GetNumTreeRCFiles());
    tree_c_files[i].resize(params.GetNumTreeRCFiles());
    tree_r_fds[i].resize(params.GetNumTreeRCFiles());
    tree_r_files[i].resize(params.GetNumTreeRCFiles());
    for (size_t j = 0; j < params.GetNumTreeRCFiles(); j++) {
      const size_t MAX = 256;
      char fname[MAX];

      // tree-c
      snprintf(fname, MAX, tree_c_filename_template, output_dir, i, j);
      remove(fname);
      tree_c_fds[i][j] = open(fname, O_RDWR | O_CREAT, (mode_t)0664);
      assert (tree_c_fds[i][j] != -1);
      lseek(tree_c_fds[i][j], tree_c_address.data_size() - 1, SEEK_SET);
      assert (write(tree_c_fds[i][j], "", 1) != -1);
      tree_c_files[i][j] = (uint8_t*)mmap(NULL, tree_c_address.data_size(),
                                          PROT_WRITE, MAP_SHARED, tree_c_fds[i][j], 0);
      if (tree_c_files[i][j] == MAP_FAILED) {
        perror("mmap failed for tree_c file");
        exit(1);
      }
      assert(madvise(tree_c_files[i][j], tree_c_address.data_size(), MADV_RANDOM) == 0);

      // tree-r
      snprintf(fname, MAX, tree_r_filename_template, output_dir, i, j);
      remove(fname);
      tree_r_fds[i][j] = open(fname, O_RDWR | O_CREAT, (mode_t)0664);
      assert (tree_r_fds[i][j] != -1);
      lseek(tree_r_fds[i][j], tree_r_address.data_size() - 1, SEEK_SET);
      assert (write(tree_r_fds[i][j], "", 1) != -1);
      tree_r_files[i][j] = (uint8_t*)mmap(NULL, tree_r_address.data_size(),
                                          PROT_WRITE, MAP_SHARED, tree_r_fds[i][j], 0);
      if (tree_r_files[i][j] == MAP_FAILED) {
        perror("mmap failed for tree_r file");
        exit(1);
      }
      assert(madvise(tree_r_files[i][j], tree_r_address.data_size(), MADV_RANDOM) == 0);
    }
  }
  
  // Compute the final offset in the file for GPU data
  tree_address_t final_tree(stream_count, TREE_ARITY, sizeof(fr_t), 0);
  final_gpu_offset_c = tree_c_address.data_size() - final_tree.data_size();
  final_gpu_offset_r = tree_r_address.data_size() - final_tree.data_size();

  // Compute an offset table used for multiple partitions
  size_t nodes_per_stream = nodes_to_read / stream_count;
  size_t layer_offset = nodes_per_stream;
  while (layer_offset >= TREE_ARITY) {
    layer_offsets_c.push_back(layer_offset);
    layer_offset /= TREE_ARITY;
  }

  layer_offset = nodes_per_stream;
  for (size_t i = 0; i < params.GetNumTreeRDiscardRows() + 1; i++) {
    layer_offset /= TREE_ARITY;
  }
  while (layer_offset >= TREE_ARITY) {
    layer_offsets_r.push_back(layer_offset);
    layer_offset /= TREE_ARITY;
  }

  // Create GPU poseidon hashers and streams
  size_t resource_id = 0;
  for (size_t i = 0; i < ngpus(); i++) {
    auto& gpu = select_gpu(i);
    poseidon_columns.push_back(new PoseidonCuda<COL_ARITY_DT>(gpu));
    poseidon_trees.push_back(new PoseidonCuda<TREE_ARITY_DT>(gpu));
      
    for (size_t j = 0; j < stream_count / ngpus(); j++) {
      resources.push_back(new gpu_resource_t<C>(params,resource_id, gpu,
                                                nodes_per_stream, batch_size,
                                                reader.alloc_node_ios()));
      resource_id++;
    }
  }

  // Register the page buffer with the CUDA driver
  size_t page_buffer_size;
  page_buffer = reader.get_buffer(page_buffer_size);
  hipHostRegister(page_buffer, page_buffer_size, hipHostRegisterDefault);

  // Set up host side buffers for returning data
  host_bufs.resize(num_host_bufs);
  host_buf_pool.create(num_host_bufs);
  host_buf_to_disk.create(num_host_bufs);
  for (size_t i = 0; i < num_host_bufs; i++) {
    host_bufs[i].data = &host_buf_storage[i * batch_size * C::PARALLEL_SECTORS];
    host_buf_pool.enqueue(&host_bufs[i]);
  }
}

template<class C>
pc2_t<C>::~pc2_t() {
  for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
    for (size_t j = 0; j < params.GetNumTreeRCFiles(); j++) {
      munmap(tree_c_files[i][j], tree_c_address.data_size());
      close(tree_c_fds[i][j]);

      munmap(tree_r_files[i][j], tree_r_address.data_size());
      close(tree_r_fds[i][j]);
    }
  }
  while (resources.size() > 0) {
    gpu_resource_t<C>* r = resources.back();
    select_gpu(r->gpu);
      
    delete r;
    resources.pop_back();
  }
  for (size_t i = 0; i < ngpus(); i++) {
    delete poseidon_columns[i];
    delete poseidon_trees[i];
  }
  hipHostUnregister(page_buffer);
}

template<class C>
void pc2_t<C>::hash() {
  auto start = chrono::high_resolution_clock::now();
  for (size_t partition = 0; partition < params.GetNumTreeRCFiles(); partition++) {
    auto pstart_gpu = chrono::high_resolution_clock::now();
    hash_gpu(partition);
    auto pstop_gpu = chrono::high_resolution_clock::now();
    hash_cpu(&tree_c_partition_roots[partition * C::PARALLEL_SECTORS],
             partition, &(gpu_results_c[0]), tree_c_files, final_gpu_offset_c);
    hash_cpu(&tree_r_partition_roots[partition * C::PARALLEL_SECTORS],
             partition, &(gpu_results_r[0]), tree_r_files, final_gpu_offset_r);
    auto pstop_cpu = chrono::high_resolution_clock::now();
    uint64_t secs_gpu = std::chrono::duration_cast<
      std::chrono::seconds>(pstop_gpu - pstart_gpu).count();
    uint64_t secs_cpu = std::chrono::duration_cast<
      std::chrono::seconds>(pstop_cpu - pstop_gpu).count();
    printf("Partition %ld took %ld seconds (gpu %ld, cpu %ld)\n",
           partition, secs_gpu + secs_cpu, secs_gpu, secs_cpu);
  }
  write_roots(&tree_c_partition_roots[0], &tree_r_partition_roots[0]);
  auto stop = chrono::high_resolution_clock::now();
  uint64_t secs = std::chrono::duration_cast<
    std::chrono::seconds>(stop - start).count();

  size_t total_page_reads = nodes_to_read * params.GetNumTreeRCFiles() /
    C::NODES_PER_PAGE * params.GetNumLayers();
  printf("pc2 took %ld seconds utilizing %0.1lf iOPS\n",
         secs, (double)total_page_reads / (double)secs);
}

template<class C>
void pc2_t<C>::hash_gpu(size_t partition) {
  assert (stream_count % ngpus() == 0);

  nodes_per_stream = nodes_to_read / stream_count;

  thread_pool_t pool(1);

  for (size_t i = 0; i < resources.size(); i++) {
    resources[i]->reset();
  }
  
  // Start a thread to process writes to disk
  atomic<bool> terminate = false;
  atomic<bool> disk_writer_done = false;
  pool.spawn([this, &terminate, &disk_writer_done]() {
    set_core_affinity(write_core);

    const size_t batch_size = 32;
    buf_to_disk_t<C>* to_disk_batch[batch_size];
    size_t count = 0;
    
    while(!terminate || host_buf_to_disk.size() > 0) {
      buf_to_disk_t<C>* to_disk = host_buf_to_disk.dequeue();
      if (to_disk != nullptr) {
        for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
          memcpy(to_disk->dst[i], to_disk->src[i], to_disk->size);
        }
        host_buf_pool.enqueue(to_disk);
      }
    }
    disk_writer_done = true;
  });
  
  bool all_done = false;
  cuda_lambda_t cuda_notify(1);
  in_ptrs_d<TREE_ARITY> in_d;
  buf_to_disk_t<C>* to_disk;
  buf_to_disk_t<C>* to_disk_r;
  fr_t* fr;

  while (!all_done) {
    all_done = true;
    for (size_t resource_num = 0; resource_num < resources.size(); resource_num++) {
      gpu_resource_t<C>& resource = *resources[resource_num];
      select_gpu(resource.gpu);
      int gpu_id = resource.gpu.id();
      fr_t* host_buf_c = (fr_t*)reader.get_buffer_id(resource.id);
      fr_t* host_buf_r = (fr_t*)reader.get_buffer_id(resources.size() + resource.id);

      if (resource.state != ResourceState::DONE) {
        all_done = false;
      }

      fr_t* out_c_d = nullptr;
      fr_t* out_r_d = nullptr;
      size_t layer_offset;
      node_id_t addr;
      size_t offset_c;
      size_t offset_r;
      uint64_t start_node;
      bool write_tree_r;

      // Device storage for the hash result
      if (resource.work_c.buf != nullptr) {
        out_c_d = &(*resource.work_c.buf)[0];
        out_r_d = &(*resource.work_r.buf)[0];
      }
      
      switch (resource.state) {
      case ResourceState::DONE:
        // Nothing
        break;
        
      case ResourceState::IDLE:
        // Initiate data read
        resource.last = !resource.scheduler_c.next([](work_item_t<gpu_buffer_t>& w) {},
                                                   &resource.work_c);
        resource.scheduler_r.next([](work_item_t<gpu_buffer_t>& w) {},
                                  &resource.work_r);
        if (resource.work_c.is_leaf) {
#ifdef DISABLE_FILE_READS
          resource.state = ResourceState::HASH_COLUMN;
          resource.column_data = reader.get_buffer_id(resource_num);
#else
          resource.state = ResourceState::DATA_READ;
#endif
        } else {
          resource.state = ResourceState::HASH_LEAF;
        }
        break;

      case ResourceState::DATA_READ:
        // Initiate the next data read
        start_node = ((uint64_t)resource.work_c.idx.node() * batch_size +
                      nodes_per_stream * resource.id +
                      partition * nodes_to_read);
        resource.column_data = reader.read_columns
          (start_node, resource.id, &resource.valid, &resource.valid_count, resource.node_ios);
        resource.state = ResourceState::DATA_WAIT;
        break;

      case ResourceState::DATA_WAIT:
        if (resource.valid.load() == resource.valid_count) {
          resource.state = ResourceState::HASH_COLUMN;
        }
        break;
      
      case ResourceState::HASH_COLUMN:
        to_disk = host_buf_pool.dequeue();
        if (to_disk == nullptr) {
          break;
        }
        
        resource.stream.HtoD(&resource.column_data_d[0], resource.column_data, resource.batch_elements);

        // Hash the columns
        poseidon_columns[gpu_id]->hash_batch_device
          (out_c_d, &resource.column_data_d[0], &resource.aux_d[0],
           batch_size * C::PARALLEL_SECTORS, C::PARALLEL_SECTORS,
           resource.stream, true, false, true, true);

        // Initiate copy of the hashed data from GPU
        fr = to_disk->data;
        resource.stream.DtoH(fr, out_c_d, batch_size * C::PARALLEL_SECTORS);

        // Initiate transfer of tree-c data to files
        layer_offset = layer_offsets_c[resource.work_c.idx.layer() - 1];
        addr = node_id_t(resource.work_c.idx.layer() - 1,
                         resource.work_c.idx.node() * batch_size + layer_offset * resource_num);
        offset_c = tree_c_address.address(addr);

        for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
          to_disk->dst[i] = (fr_t*)&tree_c_files[i][partition][offset_c];
          to_disk->src[i] = &to_disk->data[i * batch_size];
        }
        to_disk->size = batch_size * sizeof(fr_t);

        resources[resource_num]->async_done = false;
        cuda_notify.schedule(resource.stream, [this, resource_num, to_disk, offset_c]() {
          this->host_buf_to_disk.enqueue(to_disk);
          resources[resource_num]->async_done = true;
        });

        resource.state = ResourceState::HASH_COLUMN_LEAVES;
        break;
        
      case ResourceState::HASH_COLUMN_LEAVES:
        if (!resources[resource_num]->async_done) {
          break;
        }
        to_disk = host_buf_pool.dequeue();
        if (to_disk == nullptr) {
          break;
        }

        // Hash tree-c
        poseidon_trees[gpu_id]->hash_batch_device
          (out_c_d, out_c_d, &resource.aux_d[0],
           batch_size * C::PARALLEL_SECTORS / TREE_ARITY, 1,
           resource.stream, false, false, true, true);

        // Hash tree-r using layer 11
        poseidon_trees[gpu_id]->hash_batch_device
          (out_r_d,
           &resource.column_data_d[batch_size * C::PARALLEL_SECTORS * (params.GetNumLayers() - 1)],
           &resource.aux_d[0],
           batch_size * C::PARALLEL_SECTORS / TREE_ARITY,
           C::PARALLEL_SECTORS,
           resource.stream, false, true, true, true);

        // Initiate copy of the hashed data from GPU, reusing the host side column buffer
        resource.stream.DtoH(&to_disk->data[0], out_c_d,
                             batch_size * C::PARALLEL_SECTORS / TREE_ARITY);
        
        // Initiate transfer of tree-c data to files
        layer_offset = layer_offsets_c[resource.work_c.idx.layer()];
        addr = node_id_t(resource.work_c.idx.layer(),
                         resource.work_c.idx.node() * batch_size / TREE_ARITY +
                         layer_offset * resource_num);
        offset_c = tree_c_address.address(addr);
        for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
          to_disk->dst[i] = (fr_t*)&tree_c_files[i][partition][offset_c];
          to_disk->src[i] = &to_disk->data[i * TREE_ARITY];
        }
        to_disk->size = TREE_ARITY * sizeof(fr_t);
        
        resources[resource_num]->async_done = false;
        cuda_notify.schedule(resource.stream, [this, resource_num, to_disk]() {
          this->host_buf_to_disk.enqueue(to_disk);
          resources[resource_num]->async_done = true;
        });

        resource.state = ResourceState::HASH_WAIT;
        break;

      case ResourceState::HASH_LEAF:
        if (host_buf_pool.size() < 2) {
          break;
        }
        to_disk = host_buf_pool.dequeue();
        assert (to_disk != nullptr);
        
        // Hash tree-c
        for (size_t i = 0; i < TREE_ARITY; i++) {
          in_d.ptrs[i] = &(*resource.work_c.inputs[i])[0];
        }


        // TODO: cleanup
        // if (resource.id == 0 &&
        //     resource.work_c.idx.layer() == 2 &&
        //     resource.work_c.idx.node() == 0) {
        //   printf("offset_c %lx, layer %d, node %d\n",
        //          offset_c, resource.work_c.idx.layer(), resource.work_c.idx.node());
        //   resource.stream.sync();

        //   fr_t tmp[batch_size * C::PARALLEL_SECTORS];
          
        //   for (size_t i = 0; i < TREE_ARITY; i++) {
        //     resource.stream.DtoH(&tmp[0], &(*resource.work_c.inputs[i])[0],
        //                          batch_size * C::PARALLEL_SECTORS);
        //     resource.stream.sync();
        //     printf("TREE input %ld\n", i);
        //     // for (size_t batch = 0; batch < batch_size; batch++) {
        //     //   for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
        //     //     printf("%ld.%2ld.%2ld: ", i, batch, sector);
        //     //     print_buffer((uint8_t*)&tmp[batch * C::PARALLEL_SECTORS + sector], sizeof(fr_t));
        //     //   }
        //     // }
        //     for (size_t batch = 0; batch < 1; batch++) {
        //       for (size_t sector = 0; sector < 16; sector++) {
        //         printf("%ld.%2ld.%2ld: ", i, batch, sector);
        //         print_buffer((uint8_t*)&tmp[batch * C::PARALLEL_SECTORS + sector], sizeof(fr_t));
        //       }
        //     }
        //   }
        // }

        // if (resource.id != 0) {
        //   break;
        // }

        
        
        poseidon_trees[gpu_id]->hash_batch_device_ptrs
          (out_c_d, in_d, &resource.aux_d[0],
           batch_size * C::PARALLEL_SECTORS / TREE_ARITY,
           C::PARALLEL_SECTORS,
           resource.stream, false, false, true, true);

        // Hash tree-r 
        for (size_t i = 0; i < TREE_ARITY; i++) {
          in_d.ptrs[i] = &(*resource.work_r.inputs[i])[0];
        }
        poseidon_trees[gpu_id]->hash_batch_device_ptrs
          (out_r_d, in_d, &resource.aux_d[0],
           batch_size * C::PARALLEL_SECTORS / TREE_ARITY,
           C::PARALLEL_SECTORS,
           resource.stream, false, false, true, true);

        // Initiate copy of the hashed data
        resource.stream.DtoH(&to_disk->data[0], out_c_d, batch_size * C::PARALLEL_SECTORS / TREE_ARITY);
        if (resource.last) {
          // Stash the final result in a known place
          fr_t* host_buf_c = (fr_t*)reader.get_buffer_id(resource_num);
          CUDA_OK(hipMemcpyAsync(host_buf_c, &to_disk->data[0],
                                  batch_size * C::PARALLEL_SECTORS / TREE_ARITY * sizeof(fr_t),
                                  hipMemcpyHostToHost, resource.stream));
        }

        // Compute offsets in the output files - tree-c
        layer_offset = layer_offsets_c[resource.work_c.idx.layer()];
        addr = node_id_t(resource.work_c.idx.layer(),
                         resource.work_c.idx.node() * batch_size / TREE_ARITY +
                         layer_offset * resource_num);
        offset_c = tree_c_address.address(addr);
        for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
          to_disk->dst[i] = (fr_t*)&tree_c_files[i][partition][offset_c];
          to_disk->src[i] = &to_disk->data[i * TREE_ARITY];
        }
        to_disk->size = TREE_ARITY * sizeof(fr_t);


        // TODO: cleanup
        // if (resource.id == 0 &&
        //     resource.work_c.idx.layer() == 2 &&
        //     resource.work_c.idx.node() == 0) {
        //   resource.stream.sync();
        
        //   printf("TREE output\n");
        //   fr_t* hashed = &to_disk->data[0];
        //   // for (size_t batch = 0; batch < batch_size; batch++) {
        //   //   for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
        //   for (size_t batch = 0; batch < 1; batch++) {
        //     for (size_t sector = 0; sector < 16; sector++) {
        //       printf("%2ld.%2ld: ", batch, sector);
        //       print_buffer((uint8_t*)&hashed[batch * C::PARALLEL_SECTORS + sector], sizeof(fr_t));
        //     }
        //   }
        //   // for (size_t i = 0; i < batch_size; i++) {
        //   //   //out[i].from();
        //   //   printf("%2ld: ", i);
        //   //   //print_buffer((uint8_t*)&out[i], sizeof(fr_t));
        //   //   print_buffer((uint8_t*)&host_buf[i], sizeof(fr_t));
        //   //   //assert(host_buf[i] == out[i]);
        //   //   //out[i].to();
        //   // }
        //   //exit(1);
        // }

        
        // tree-r
        write_tree_r = resource.work_r.idx.layer() > params.GetNumTreeRDiscardRows();
        if (write_tree_r) {
          to_disk_r = host_buf_pool.dequeue();
          assert (to_disk_r != nullptr);
          resource.stream.DtoH(&to_disk_r->data[0], out_r_d,
                               batch_size * C::PARALLEL_SECTORS / TREE_ARITY);
          if (resource.last) {
            // Stash the final result in a known place
            fr_t* host_buf_r = (fr_t*)reader.get_buffer_id(resources.size() + resource_num);
            CUDA_OK(hipMemcpyAsync(host_buf_r, &to_disk_r->data[0],
                                    batch_size * C::PARALLEL_SECTORS / TREE_ARITY * sizeof(fr_t),
                                    hipMemcpyHostToHost, resource.stream));
          }

          layer_offset = layer_offsets_r[resource.work_r.idx.layer() - params.GetNumTreeRDiscardRows() - 1];
          addr = node_id_t(resource.work_r.idx.layer() - params.GetNumTreeRDiscardRows() - 1,
                           resource.work_r.idx.node() * batch_size / TREE_ARITY +
                           layer_offset * resource_num);
          offset_r = tree_r_address.address(addr);
          for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
            to_disk_r->dst[i] = (fr_t*)&tree_r_files[i][partition][offset_r];
            to_disk_r->src[i] = &to_disk_r->data[i * TREE_ARITY];
          }
          to_disk_r->size = TREE_ARITY * sizeof(fr_t);
        }
        
        // Initiate transfer of data to files
        resources[resource_num]->async_done = false;
        cuda_notify.schedule(resource.stream, [this, resource_num,
                                               to_disk, to_disk_r, write_tree_r]() {
          this->host_buf_to_disk.enqueue(to_disk);
          if (write_tree_r) {
            this->host_buf_to_disk.enqueue(to_disk_r);
          }
          resources[resource_num]->async_done = true;
        });

        resource.state = ResourceState::HASH_WAIT;
        break;
      
      case ResourceState::HASH_WAIT:
        if (resource.async_done.load() == true) {
          if (resource.last) {
            resource.state = ResourceState::DONE;
          } else {
            resource.state = ResourceState::IDLE;
          }
        }
        break;

      default:
        abort();
      }
    }
  }
  //printf("PC2: GPU state machine done, syncing streams\n");
  for (size_t resource_num = 0; resource_num < stream_count; resource_num++) {
    resources[resource_num]->stream.sync();
  }

  terminate = true;

  // Cache the final portions of the tree for CPU hashing
  //printf("PC2: GPU state machine done, caching final nodes\n");
  size_t stride = batch_size * C::PARALLEL_SECTORS / TREE_ARITY;
  for (size_t resource_num = 0; resource_num < stream_count; resource_num++) {
    fr_t* host_buf_c = (fr_t*)reader.get_buffer_id(resource_num);
    memcpy(&gpu_results_c[resource_num * stride],
           &host_buf_c[0], batch_size * C::PARALLEL_SECTORS / TREE_ARITY * sizeof(fr_t));
  }
  for (size_t resource_num = 0; resource_num < stream_count; resource_num++) {
    fr_t* host_buf_r = (fr_t*)reader.get_buffer_id(resources.size() + resource_num);
    memcpy(&gpu_results_r[resource_num * stride],
           &host_buf_r[0], batch_size * C::PARALLEL_SECTORS / TREE_ARITY * sizeof(fr_t));
  }

  // Really only need this at the last partition...
  //printf("PC2: GPU state machine done, waiting for disk writer to complete\n");
  while (!disk_writer_done) {}
}

template<class C>
void pc2_t<C>::hash_cpu(fr_t* roots, size_t partition, fr_t* input,
                        vector<uint8_t*> tree_files[C::PARALLEL_SECTORS],
                        size_t file_offset) {
  const size_t nodes_to_hash = stream_count; // Number of GPU streams
  
  tree_address_t final_tree(nodes_to_hash, TREE_ARITY, sizeof(fr_t), 0);

  Poseidon hasher(TREE_ARITY);

  auto hash_func = [this, &hasher, &final_tree, input, partition, tree_files, file_offset]
    (work_item_t<host_buffer_t>& w) {
    node_id_t addr(w.idx.layer() - 1, w.idx.node());
    size_t offset = final_tree.address(addr) + file_offset;

    if (w.is_leaf) {
      for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
        fr_t* out = &(*w.buf)[sector];
        fr_t in[TREE_ARITY];
        
        for (size_t i = 0; i < TREE_ARITY; i++) {
          in[i] = input[w.idx.node() * TREE_ARITY * C::PARALLEL_SECTORS +
                        sector * TREE_ARITY + i];
        }
        hasher.Hash((uint8_t*)out, (uint8_t*)in);
        memcpy(&tree_files[sector][partition][offset],
               &out[0], sizeof(fr_t));
      }
    } else {
      for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
        fr_t* out = &(*w.buf)[sector];
        fr_t in[TREE_ARITY];
        for (size_t i = 0; i < TREE_ARITY; i++) {
          in[i] = (*w.inputs[i])[sector];
        }
        hasher.Hash((uint8_t*)out, (uint8_t*)in);
        
        memcpy(&tree_files[sector][partition][offset],
               &out[0], sizeof(fr_t));
      }
    }
  };
  
  buffers_t<host_buffer_t> buffers(C::PARALLEL_SECTORS);
  scheduler_t<host_buffer_t> scheduler(nodes_to_hash, TREE_ARITY, buffers);
  host_buffer_t* host_buf = scheduler.run(hash_func);
  memcpy(roots, &(*host_buf)[0], sizeof(fr_t) * C::PARALLEL_SECTORS);
}

template<class C>
void pc2_t<C>::write_roots(fr_t* roots_c, fr_t* roots_r) {
  if (params.GetNumTreeRCFiles() > 1) {
    Poseidon hasher(TREE_ARITY);
    for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
      fr_t out_c;
      fr_t in[TREE_ARITY];
      for (size_t i = 0; i < TREE_ARITY; i++) {
        in[i] = roots_c[i * C::PARALLEL_SECTORS + sector];
      }
      hasher.Hash((uint8_t*)&out_c, (uint8_t*)in);

      fr_t out_r;
      for (size_t i = 0; i < TREE_ARITY; i++) {
        in[i] = roots_r[i * C::PARALLEL_SECTORS + sector];
      }
      hasher.Hash((uint8_t*)&out_r, (uint8_t*)in);
    
      const size_t MAX = 256;
      char fname[MAX];
      snprintf(fname, MAX, p_aux_template, output_dir, sector);
      int p_aux = open(fname, O_RDWR | O_CREAT, (mode_t)0664);
      assert (p_aux != -1);
      assert (write(p_aux, &out_c, sizeof(fr_t)) == sizeof(fr_t));
      assert (write(p_aux, &out_r, sizeof(fr_t)) == sizeof(fr_t));
      close(p_aux);
    }
  } else {
    for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
      fr_t out_c = roots_c[sector];
      fr_t out_r = roots_r[sector];

      const size_t MAX = 256;
      char fname[MAX];
      snprintf(fname, MAX, p_aux_template, output_dir, sector);
      int p_aux = open(fname, O_RDWR | O_CREAT, (mode_t)0664);
      assert (p_aux != -1);
      assert (write(p_aux, &out_c, sizeof(fr_t)) == sizeof(fr_t));
      assert (write(p_aux, &out_r, sizeof(fr_t)) == sizeof(fr_t));
      close(p_aux);
    }
  }
}

template<class C>
void pc2_hash(SectorParameters& params, column_reader_t<C>& reader,
              size_t nodes_to_read, size_t batch_size,
              size_t stream_count, int write_core, const char* output_dir) {
  pc2_t<C> pc2(params, reader, nodes_to_read, batch_size, stream_count, write_core, output_dir);
  pc2.hash();
}

template void pc2_hash<sealing_config128_t>(SectorParameters& params,
                                            column_reader_t<sealing_config128_t>& reader,
                                            size_t nodes_to_read, size_t batch_size,
                                            size_t stream_count, int write_core,
                                            const char* output_dir);
template void pc2_hash<sealing_config64_t>(SectorParameters& params,
                                           column_reader_t<sealing_config64_t>& reader,
                                           size_t nodes_to_read, size_t batch_size,
                                           size_t stream_count, int write_core,
                                           const char* output_dir);
template void pc2_hash<sealing_config32_t>(SectorParameters& params,
                                           column_reader_t<sealing_config32_t>& reader,
                                           size_t nodes_to_read, size_t batch_size,
                                           size_t stream_count, int write_core,
                                           const char* output_dir);
template void pc2_hash<sealing_config16_t>(SectorParameters& params,
                                           column_reader_t<sealing_config16_t>& reader,
                                           size_t nodes_to_read, size_t batch_size,
                                           size_t stream_count, int write_core,
                                           const char* output_dir);
template void pc2_hash<sealing_config8_t>(SectorParameters& params,
                                          column_reader_t<sealing_config8_t>& reader,
                                          size_t nodes_to_read, size_t batch_size,
                                          size_t stream_count, int write_core,
                                          const char* output_dir);
template void pc2_hash<sealing_config4_t>(SectorParameters& params,
                                          column_reader_t<sealing_config4_t>& reader,
                                          size_t nodes_to_read, size_t batch_size,
                                          size_t stream_count, int write_core,
                                          const char* output_dir);
template void pc2_hash<sealing_config2_t>(SectorParameters& params,
                                          column_reader_t<sealing_config2_t>& reader,
                                          size_t nodes_to_read, size_t batch_size,
                                          size_t stream_count, int write_core,
                                          const char* output_dir);

#endif
