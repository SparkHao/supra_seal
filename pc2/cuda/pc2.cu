// Copyright Supranational LLC

#include "../../poseidon/cuda/poseidon.cu"
#include "../../util/debug_helpers.hpp"
#include "host_ptr_t.hpp"

#ifndef __CUDA_ARCH__

#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/mman.h>

#include <filesystem>
#include <chrono>
#include "../planner.cpp"
#include "pc2.cuh"
#include "cuda_lambda_t.hpp"
#include "../../util/util.hpp"

template<class C>
pc2_t<C>::pc2_t(SectorParameters& _params, topology_t& _topology,
                bool _tree_r_only, streaming_node_reader_t<C>& _reader,
                size_t _nodes_to_read, size_t _batch_size,
                size_t _stream_count,
                const char** _data_filenames, const char* _output_dir) :
  params(_params),
  topology(_topology),
  tree_r_only(_tree_r_only),
  reader(_reader),
  nodes_to_read(_nodes_to_read),
  batch_size(_batch_size),
  tree_c_address(params.GetNumNodes() / params.GetNumTreeRCFiles(),
                 TREE_ARITY, NODE_SIZE, 0),
  tree_r_address(params.GetNumNodes() / params.GetNumTreeRCFiles(),
                 TREE_ARITY, NODE_SIZE, params.GetNumTreeRDiscardRows() + 1),
  stream_count(_stream_count),
  tree_c_partition_roots(C::PARALLEL_SECTORS * TREE_ARITY),
  tree_r_partition_roots(C::PARALLEL_SECTORS * TREE_ARITY),
  gpu_results_c(tree_r_only ? 0 :_batch_size * C::PARALLEL_SECTORS / TREE_ARITY * stream_count),
  gpu_results_r(_batch_size * C::PARALLEL_SECTORS / TREE_ARITY * stream_count),
  host_buf_storage(num_host_bufs * batch_size * C::PARALLEL_SECTORS * 2),
  data_filenames(_data_filenames),
  output_dir(_output_dir)
{
  assert (TREE_ARITY == params.GetNumTreeRCArity());
  assert (nodes_to_read % stream_count == 0);

  // Put layer11 / sealed file in a replicas directory if it exists
  std::string pc2_replica_output_dir = output_dir;
  pc2_replica_output_dir += "/replicas";
  if (!std::filesystem::exists(pc2_replica_output_dir.c_str())) {
    pc2_replica_output_dir = output_dir;
  }

  
  if (C::PARALLEL_SECTORS == 1) {
    p_aux_template = "%s/p_aux";
  } else {
    p_aux_template = "%s/%03ld/p_aux";
  }
  // Open all tree-c and tree-r files
  const char* tree_c_filename_template;
  const char* tree_r_filename_template;
  if (C::PARALLEL_SECTORS == 1) {
    if (params.GetNumTreeRCFiles() > 1) {
      tree_c_filename_template = "%s/sc-02-data-tree-c-%ld.dat";
      tree_r_filename_template = "%s/sc-02-data-tree-r-last-%ld.dat";
    } else {
      tree_c_filename_template = "%s/sc-02-data-tree-c.dat";
      tree_r_filename_template = "%s/sc-02-data-tree-r-last.dat";
    }
  } else {
    if (params.GetNumTreeRCFiles() > 1) {
      tree_c_filename_template = "%s/%03ld/sc-02-data-tree-c-%ld.dat";
      tree_r_filename_template = "%s/%03ld/sc-02-data-tree-r-last-%ld.dat";
    } else {
      tree_c_filename_template = "%s/%03ld/sc-02-data-tree-c.dat";
      tree_r_filename_template = "%s/%03ld/sc-02-data-tree-r-last.dat";
    }
  }
  // And sealed files
  const char* sealed_filename_template = "%s/%03ld/sealed-file";
  const char* layer11_filename_template = "%s/%03ld/layer11-file";

  if (!std::filesystem::exists(output_dir)) {
    std::filesystem::create_directory(output_dir);
  }
  has_cc_sectors = false;
  has_non_cc_sectors = false;
  
  const size_t MAX = 256;
  char fname[MAX];
  for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
    // Create sector subdirs
    if (C::PARALLEL_SECTORS == 1) {
      snprintf(fname, MAX, "%s", output_dir);
    } else {
      snprintf(fname, MAX, "%s/%03ld", output_dir, i);
    }
    if (!std::filesystem::exists(fname)) {
      std::filesystem::create_directory(fname);
    }
    if (!tree_r_only) {
      if (C::PARALLEL_SECTORS == 1) {
        snprintf(fname, MAX, "%s", pc2_replica_output_dir.c_str());
      } else {
        snprintf(fname, MAX, "%s/%03ld", pc2_replica_output_dir.c_str(), i);
      }
      if (!std::filesystem::exists(fname)) {
        std::filesystem::create_directory(fname);
      }
    }

    if (!tree_r_only) {
      tree_c_files[i].resize(params.GetNumTreeRCFiles());
    }
    tree_r_files[i].resize(params.GetNumTreeRCFiles());
    for (size_t j = 0; j < params.GetNumTreeRCFiles(); j++) {
      // tree-c
      if (!tree_r_only) {
        if (C::PARALLEL_SECTORS == 1) {
          if (params.GetNumTreeRCFiles() > 1) {
            snprintf(fname, MAX, tree_c_filename_template, output_dir, j);
          } else {
            snprintf(fname, MAX, tree_c_filename_template, output_dir);
          }
        } else {
          if (params.GetNumTreeRCFiles() > 1) {
            snprintf(fname, MAX, tree_c_filename_template, output_dir, i, j);
          } else {
            snprintf(fname, MAX, tree_c_filename_template, output_dir, i);
          }
        }
        assert(tree_c_files[i][j].mmap_write(fname, tree_c_address.data_size(), true) == 0);
        tree_c_files[i][j].advise_random();
      }
                            
      // tree-r
      if (C::PARALLEL_SECTORS == 1) {
        if (params.GetNumTreeRCFiles() > 1) {
          snprintf(fname, MAX, tree_r_filename_template, output_dir, j);
        } else {
          snprintf(fname, MAX, tree_r_filename_template, output_dir);
        }
      } else {
        if (params.GetNumTreeRCFiles() > 1) {
          snprintf(fname, MAX, tree_r_filename_template, output_dir, i, j);
        } else {
          snprintf(fname, MAX, tree_r_filename_template, output_dir, i);
        }
      }
      assert(tree_r_files[i][j].mmap_write(fname, tree_r_address.data_size(), true) == 0);
      tree_r_files[i][j].advise_random();
    }

    // Data files for encoding
    if (!tree_r_only) {
      if (data_filenames != nullptr && data_filenames[i] != nullptr) {
        data_files[i].mmap_read(data_filenames[i], SECTOR_SIZE);
        // If there is a data file present we will encode layer 11 and write the
        // sealed data
        snprintf(fname, MAX, sealed_filename_template, pc2_replica_output_dir.c_str(), i);
        assert(sealed_files[i].mmap_write(fname, SECTOR_SIZE, true) == 0);
        has_non_cc_sectors = true;
      } else {
        // Write the raw layer 11 data
        // TODO: no way to differentiate cc vs remote data
        //snprintf(fname, MAX, layer11_filename_template, pc2_replica_output_dir.c_str(), i);
        snprintf(fname, MAX, sealed_filename_template, pc2_replica_output_dir.c_str(), i);
        assert(sealed_files[i].mmap_write(fname, SECTOR_SIZE, true) == 0);
        has_cc_sectors = true;
      }
    }
  }
  
  // Compute the final offset in the file for GPU data
  tree_address_t final_tree(batch_size, TREE_ARITY, sizeof(fr_t), 0);
  final_gpu_offset_c = tree_c_address.data_size() - final_tree.data_size();
  final_gpu_offset_r = tree_r_address.data_size() - final_tree.data_size();

  // Compute an offset table used for multiple partitions
  size_t nodes_per_stream = nodes_to_read / stream_count;
  size_t layer_offset = nodes_per_stream;
  while (layer_offset >= TREE_ARITY) {
    layer_offsets_c.push_back(layer_offset);
    layer_offset /= TREE_ARITY;
  }

  layer_offset = nodes_per_stream;
  for (size_t i = 0; i < params.GetNumTreeRDiscardRows() + 1; i++) {
    layer_offset /= TREE_ARITY;
  }
  while (layer_offset >= TREE_ARITY) {
    layer_offsets_r.push_back(layer_offset);
    layer_offset /= TREE_ARITY;
  }

  // Create GPU poseidon hashers and streams
  size_t resource_id = 0;
  for (size_t i = 0; i < ngpus(); i++) {
    auto& gpu = select_gpu(i);
    if (!tree_r_only) {
      poseidon_columns.push_back(new PoseidonCuda<COL_ARITY_DT>(gpu));
    }
    poseidon_trees.push_back(new PoseidonCuda<TREE_ARITY_DT>(gpu));
      
    for (size_t j = 0; j < stream_count / ngpus(); j++) {
      resources.push_back(new gpu_resource_t<C>(params,resource_id, gpu,
                                                nodes_per_stream, batch_size));
      resource_id++;
    }
  }

  // Register the page buffer with the CUDA driver
  size_t page_buffer_size = 0;
  page_buffer = reader.get_full_buffer(page_buffer_size);
  hipHostRegister(page_buffer, page_buffer_size, hipHostRegisterDefault);

  // Set up host side buffers for returning data
  host_bufs0.resize(num_host_bufs);
  host_buf_pool0.create(num_host_bufs);
  host_buf_to_disk0.create(num_host_bufs);

  if (!tree_r_only) {
    host_bufs1.resize(num_host_bufs);
    host_buf_pool1.create(num_host_bufs);
    host_buf_to_disk1.create(num_host_bufs);
  }
  for (size_t i = 0; i < num_host_bufs; i++) {
    host_bufs0[i].data = &host_buf_storage[i * batch_size * C::PARALLEL_SECTORS];
    host_buf_pool0.enqueue(&host_bufs0[i]);
    
    if (!tree_r_only) {
      host_bufs1[i].data = &host_buf_storage[num_host_bufs * batch_size * C::PARALLEL_SECTORS +
                                             i * batch_size * C::PARALLEL_SECTORS];
      host_buf_pool1.enqueue(&host_bufs1[i]);
    }
  }
}

template<class C>
pc2_t<C>::~pc2_t() {
  while (resources.size() > 0) {
    gpu_resource_t<C>* r = resources.back();
    select_gpu(r->gpu);
      
    delete r;
    resources.pop_back();
  }
  for (size_t i = 0; i < ngpus(); i++) {
    if (!tree_r_only) {
      delete poseidon_columns[i];
    }
    delete poseidon_trees[i];
  }
  hipHostUnregister(page_buffer);
}

template<class C>
void pc2_t<C>::hash() {
  auto start = std::chrono::high_resolution_clock::now();
  for (size_t partition = 0; partition < params.GetNumTreeRCFiles(); partition++) {
    auto pstart_gpu = std::chrono::high_resolution_clock::now();
    hash_gpu(partition);
    auto pstop_gpu = std::chrono::high_resolution_clock::now();
    if (!tree_r_only) {
      hash_cpu(&tree_c_partition_roots[partition * C::PARALLEL_SECTORS],
               partition, &(gpu_results_c[0]), tree_c_files, final_gpu_offset_c);
    }
    hash_cpu(&tree_r_partition_roots[partition * C::PARALLEL_SECTORS],
             partition, &(gpu_results_r[0]), tree_r_files, final_gpu_offset_r);
    auto pstop_cpu = std::chrono::high_resolution_clock::now();
    uint64_t secs_gpu = std::chrono::duration_cast<
      std::chrono::seconds>(pstop_gpu - pstart_gpu).count();
    uint64_t secs_cpu = std::chrono::duration_cast<
      std::chrono::seconds>(pstop_cpu - pstop_gpu).count();
    printf("Partition %ld took %ld seconds (gpu %ld, cpu %ld)\n",
           partition, secs_gpu + secs_cpu, secs_gpu, secs_cpu);
  }
  write_roots(&tree_c_partition_roots[0], &tree_r_partition_roots[0]);
  auto stop = std::chrono::high_resolution_clock::now();
  uint64_t secs = std::chrono::duration_cast<
    std::chrono::seconds>(stop - start).count();

  size_t total_page_reads = nodes_to_read * params.GetNumTreeRCFiles() /
    C::NODES_PER_PAGE * params.GetNumLayers();
  printf("pc2 took %ld seconds utilizing %0.1lf iOPS\n",
         secs, (double)total_page_reads / (double)secs);
}

template<class C>
void pc2_t<C>::process_writes(int core,
                              mt_fifo_t<buf_to_disk_t<C>>& to_disk_fifo,
                              mt_fifo_t<buf_to_disk_t<C>>& pool,
                              std::atomic<bool>& terminate,
                              std::atomic<int>& disk_writer_done) {
  set_core_affinity(core);
  
  while(!terminate || to_disk_fifo.size() > 0) {
    if (pool.is_full()) {
      continue;
    }

    buf_to_disk_t<C>* to_disk = to_disk_fifo.dequeue();
    if (to_disk != nullptr) {
#ifndef DISABLE_FILE_WRITES
      if (to_disk->stride == 1) {
        // Copy chunks of contiguous data
        for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
          if (to_disk->src[i] != nullptr) {
            memcpy(to_disk->dst[i], to_disk->src[i], to_disk->size * sizeof(fr_t));
          }
        }
      } else {
        //  Copy strided src data
        for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
          if (to_disk->src[i] != nullptr) {
            for (size_t j = 0; j < to_disk->size; j++) {
              fr_t elmt = to_disk->src[i][j * to_disk->stride];
              if (to_disk->reverse) {
                node_t* n = (node_t*)&elmt;
                n->reverse_l();
              }
              to_disk->dst[i][j] = elmt;
            }
          }
        }
      }
#endif
      pool.enqueue(to_disk);
    }
  }
  disk_writer_done++;
}

template<class C>
void pc2_t<C>::hash_gpu(size_t partition) {
  assert (stream_count % ngpus() == 0);

  nodes_per_stream = nodes_to_read / stream_count;

  thread_pool_t pool(2);

  for (size_t i = 0; i < resources.size(); i++) {
    resources[i]->reset();
  }
  
  // Start a thread to process writes to disk
  std::atomic<bool> terminate = false;
  std::atomic<int> disk_writer_done(0);
  // tree-c and tree-r
  pool.spawn([this, &terminate, &disk_writer_done]() {
    process_writes(this->topology.pc2_writer0,
                   host_buf_to_disk0, host_buf_pool0,
                   terminate, disk_writer_done);
  });
  // last layer / sealed file
  if (!tree_r_only) {
    pool.spawn([this, &terminate, &disk_writer_done]() {
      process_writes(this->topology.pc2_writer1,
                     host_buf_to_disk1, host_buf_pool1,
                     terminate, disk_writer_done);
    });
  }
  
  bool all_done = false;
  cuda_lambda_t cuda_notify(1);
  in_ptrs_d<TREE_ARITY> in_d;
  buf_to_disk_t<C>* to_disk = nullptr;
  buf_to_disk_t<C>* to_disk_r = nullptr;
  fr_t* fr = nullptr;
  size_t disk_bufs_needed = 0;
  
  while (!all_done) {
    all_done = true;
    for (size_t resource_num = 0; resource_num < resources.size(); resource_num++) {
      gpu_resource_t<C>& resource = *resources[resource_num];
      select_gpu(resource.gpu);
      int gpu_id = resource.gpu.id();

      if (resource.state != ResourceState::DONE) {
        all_done = false;
      }

      fr_t* out_c_d = nullptr;
      fr_t* out_r_d = nullptr;
      size_t layer_offset;
      node_id_t addr;
      size_t offset_c;
      size_t offset_r;
      bool write_tree_r;

      // Device storage for the hash result
      if (resource.work_c.buf != nullptr) {
        out_c_d = &(*resource.work_c.buf)[0];
        out_r_d = &(*resource.work_r.buf)[0];
      }

      switch (resource.state) {
      case ResourceState::DONE:
        // Nothing
        break;
        
      case ResourceState::IDLE:
        // Initiate data read
        resource.last = !resource.scheduler_c.next([](work_item_t<gpu_buffer_t>& w) {},
                                                   &resource.work_c);
        resource.scheduler_r.next([](work_item_t<gpu_buffer_t>& w) {},
                                  &resource.work_r);
        if (resource.work_c.is_leaf) {
#ifdef DISABLE_FILE_READS
          resource.state = ResourceState::HASH_COLUMN;
#else
          resource.state = ResourceState::DATA_READ;
#endif
        } else {
          resource.state = ResourceState::HASH_LEAF;
        }
        break;

      case ResourceState::DATA_READ:
        // Initiate the next data read
        resource.start_node = (// Perform batch_size nodes in parallel
                               (uint64_t)resource.work_c.idx.node() * batch_size +
                               // Each resource (GPU stream) works on a differet nodes_per_stream chunk
                               nodes_per_stream * resource.id +
                               // Each partition is size nodes_to_read
                               partition * nodes_to_read);
        resource.column_data = (fr_t*)reader.load_layers
          (resource.id,
           tree_r_only ? params.GetNumLayers() - 1 : 0, // start layer
           resource.start_node, batch_size,
           tree_r_only ? 1 : params.GetNumLayers(), // num_layers
           &resource.valid, &resource.valid_count);
        resource.state = ResourceState::DATA_WAIT;
        break;

      case ResourceState::DATA_WAIT:
        if (resource.valid.load() == resource.valid_count) {
          if (!tree_r_only) {
            // Prepare to Write layer 11 / sealed data to disk          
            if (host_buf_to_disk1.is_full()) {
              break;
            }
            to_disk = host_buf_pool1.dequeue();
            if (to_disk == nullptr) {
              break;
            }
          }
          fr_t* encode_buf = &resource.replica_data[0];
          
          // Copy layer 11 data to to_disk buffer for encoding/writing
          // If only building tree-r then only the last layer is present
          fr_t* layer11;
          if (tree_r_only) {
            layer11 = &resource.column_data[0];
          } else {
            layer11 = &resource.column_data[C::PARALLEL_SECTORS *
                                            (params.GetNumLayers() - 1) * batch_size];
          }
          memcpy(encode_buf, layer11,
                 C::PARALLEL_SECTORS * batch_size * sizeof(fr_t));

          // Encode non CC sectors
          for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
            if (data_files[i].is_open()) {
              for (size_t j = 0; j < batch_size; j++) {
                // Perform the field add without moving to Montgomery space
                fr_t data = data_files[i][resource.start_node + j];
                fr_t* elmt = &encode_buf[i + j * C::PARALLEL_SECTORS];
                node_t* n = (node_t*)elmt;
                n->reverse_l();
                *elmt += data;
                n->reverse_l();
              }
            }
          }

          if (!tree_r_only) {
            // Prepare write pointers
            to_disk->size = batch_size;
            to_disk->stride = C::PARALLEL_SECTORS;
            to_disk->reverse = true;
            for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
              to_disk->src[i] = &to_disk->data[i];
              to_disk->dst[i] = &sealed_files[i][resource.start_node];
              if (data_files[i].is_open()) {
              }
            }
            // Copy the encoded replica data into the disk buffer
            memcpy(&to_disk->data[0],
                   &resource.replica_data[0],
                   batch_size * C::PARALLEL_SECTORS * sizeof(fr_t));
          }
          
          if (tree_r_only) {
            resource.state = ResourceState::HASH_COLUMN_LEAVES;
          } else {
            host_buf_to_disk1.enqueue(to_disk);
            resource.state = ResourceState::HASH_COLUMN;
          }
        }
        break;
      
      case ResourceState::HASH_COLUMN:
        if (host_buf_to_disk0.is_full()) {
          break;
        }
        to_disk = host_buf_pool0.dequeue();
        if (to_disk == nullptr) {
          break;
        }
        
        resource.stream.HtoD(&resource.column_data_d[0], resource.column_data, resource.batch_elements);

        // Hash the columns
        poseidon_columns[gpu_id]->hash_batch_device
          (out_c_d, &resource.column_data_d[0], &resource.aux_d[0],
           batch_size * C::PARALLEL_SECTORS, C::PARALLEL_SECTORS,
           resource.stream, true, false, true, true,
           !reader.data_is_big_endian());

        // Initiate copy of the hashed data from GPU
        fr = to_disk->data;
        resource.stream.DtoH(fr, out_c_d, batch_size * C::PARALLEL_SECTORS);

        // Initiate transfer of tree-c data to files
        layer_offset = layer_offsets_c[resource.work_c.idx.layer() - 1];
        addr = node_id_t(resource.work_c.idx.layer() - 1,
                         resource.work_c.idx.node() * batch_size + layer_offset * resource_num);
        offset_c = tree_c_address.address(addr);

        for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
          to_disk->dst[i] = (fr_t*)&tree_c_files[i][partition][offset_c];
          to_disk->src[i] = &to_disk->data[i * batch_size];
        }
        to_disk->size = batch_size;
        to_disk->stride = 1;
        to_disk->reverse = false;

        resources[resource_num]->async_done = false;
        cuda_notify.schedule(resource.stream, [this, resource_num, to_disk, offset_c]() {
          this->host_buf_to_disk0.enqueue(to_disk);
          resources[resource_num]->async_done = true;
        });

        resource.state = ResourceState::HASH_COLUMN_LEAVES;
        break;
        
      case ResourceState::HASH_COLUMN_LEAVES:
        if (!resources[resource_num]->async_done) {
          break;
        }
        if (!tree_r_only) {
          if (host_buf_to_disk0.is_full()) {
            break;
          }
          to_disk = host_buf_pool0.dequeue();
          if (to_disk == nullptr) {
            break;
          }

          // Hash tree-c
          poseidon_trees[gpu_id]->hash_batch_device
            (out_c_d, out_c_d, &resource.aux_d[0],
             batch_size * C::PARALLEL_SECTORS / TREE_ARITY, 1,
             resource.stream, false, false, true, true,
             !reader.data_is_big_endian());
        }

        // Hash tree-r using the replica data. If there are any non-CC
        // sectors then copy the encoded replica data over
        if (has_non_cc_sectors || tree_r_only) {
          resource.stream.HtoD
            (&resource.column_data_d[batch_size * C::PARALLEL_SECTORS * (params.GetNumLayers() - 1)],
             &resource.replica_data[0], C::PARALLEL_SECTORS * batch_size);
        }
        poseidon_trees[gpu_id]->hash_batch_device
          (out_r_d,
           &resource.column_data_d[batch_size * C::PARALLEL_SECTORS * (params.GetNumLayers() - 1)],
           &resource.aux_d[0],
           batch_size * C::PARALLEL_SECTORS / TREE_ARITY,
           C::PARALLEL_SECTORS,
           resource.stream, false, true, true, true,
           !reader.data_is_big_endian());

        if (!tree_r_only) {
          // Initiate copy of the hashed data from GPU, reusing the host side column buffer
          resource.stream.DtoH(&to_disk->data[0], out_c_d,
                               batch_size * C::PARALLEL_SECTORS / TREE_ARITY);
          
          // Initiate transfer of tree-c data to files
          layer_offset = layer_offsets_c[resource.work_c.idx.layer()];
          addr = node_id_t(resource.work_c.idx.layer(),
                           resource.work_c.idx.node() * batch_size / TREE_ARITY +
                           layer_offset * resource_num);
          offset_c = tree_c_address.address(addr);
          for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
            to_disk->dst[i] = (fr_t*)&tree_c_files[i][partition][offset_c];
            to_disk->src[i] = &to_disk->data[i * batch_size / TREE_ARITY];
          }
          to_disk->size = batch_size / TREE_ARITY;
          to_disk->stride = 1;
          to_disk->reverse = false;
        }
        
        resources[resource_num]->async_done = false;
        cuda_notify.schedule(resource.stream, [this, resource_num, to_disk]() {
          if (!tree_r_only) {
            this->host_buf_to_disk0.enqueue(to_disk);
          }
          resources[resource_num]->async_done = true;
        });
        
        resource.state = ResourceState::HASH_WAIT;
        break;

      case ResourceState::HASH_LEAF:
        disk_bufs_needed = tree_r_only ? 1 : 2;
        if (host_buf_to_disk0.free_count() < disk_bufs_needed) {
          break;
        }
        if (host_buf_pool0.size() < disk_bufs_needed) {
          break;
        }
        if (!tree_r_only) {
          to_disk = host_buf_pool0.dequeue();
          assert (to_disk != nullptr);
        
          // Hash tree-c
          for (size_t i = 0; i < TREE_ARITY; i++) {
            in_d.ptrs[i] = &(*resource.work_c.inputs[i])[0];
          }

          poseidon_trees[gpu_id]->hash_batch_device_ptrs
            (out_c_d, in_d, &resource.aux_d[0],
             batch_size * C::PARALLEL_SECTORS / TREE_ARITY,
             C::PARALLEL_SECTORS,
             resource.stream, false, false, true, true,
             !reader.data_is_big_endian());
        }
        
        // Hash tree-r 
        for (size_t i = 0; i < TREE_ARITY; i++) {
          in_d.ptrs[i] = &(*resource.work_r.inputs[i])[0];
        }
        poseidon_trees[gpu_id]->hash_batch_device_ptrs
          (out_r_d, in_d, &resource.aux_d[0],
           batch_size * C::PARALLEL_SECTORS / TREE_ARITY,
           C::PARALLEL_SECTORS,
           resource.stream, false, false, true, true,
           !reader.data_is_big_endian());
        
        if (!tree_r_only) {
          // Initiate copy of the hashed data
          resource.stream.DtoH(&to_disk->data[0], out_c_d,
                               batch_size * C::PARALLEL_SECTORS / TREE_ARITY);
          if (resource.last) {
            // Stash the final result in a known place
            size_t stride = batch_size * C::PARALLEL_SECTORS / TREE_ARITY;
            fr_t* host_buf_c = (fr_t*)&gpu_results_c[resource.id * stride];
            CUDA_OK(hipMemcpyAsync(host_buf_c, &to_disk->data[0],
                                    batch_size * C::PARALLEL_SECTORS / TREE_ARITY * sizeof(fr_t),
                                    hipMemcpyHostToHost, resource.stream));
          }

          // Compute offsets in the output files - tree-c
          layer_offset = layer_offsets_c[resource.work_c.idx.layer()];
          addr = node_id_t(resource.work_c.idx.layer(),
                           resource.work_c.idx.node() * batch_size / TREE_ARITY +
                           layer_offset * resource_num);
          offset_c = tree_c_address.address(addr);
          for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
            to_disk->dst[i] = (fr_t*)&tree_c_files[i][partition][offset_c];
            to_disk->src[i] = &to_disk->data[i * batch_size / TREE_ARITY];
          }
          to_disk->size = batch_size / TREE_ARITY;
          to_disk->stride = 1;
          to_disk->reverse = false;
        }
        
        // tree-r
        write_tree_r = resource.work_r.idx.layer() > params.GetNumTreeRDiscardRows();
        if (write_tree_r) {
          to_disk_r = host_buf_pool0.dequeue();
          assert (to_disk_r != nullptr);
          resource.stream.DtoH(&to_disk_r->data[0], out_r_d,
                               batch_size * C::PARALLEL_SECTORS / TREE_ARITY);
          
          if (resource.last) {
            // Stash the final result in a known place
            size_t stride = batch_size * C::PARALLEL_SECTORS / TREE_ARITY;
            fr_t* host_buf_r = (fr_t*)&gpu_results_r[resource.id * stride];
            CUDA_OK(hipMemcpyAsync(host_buf_r, &to_disk_r->data[0],
                                    batch_size * C::PARALLEL_SECTORS / TREE_ARITY * sizeof(fr_t),
                                    hipMemcpyHostToHost, resource.stream));
          }

          layer_offset = layer_offsets_r[resource.work_r.idx.layer() - params.GetNumTreeRDiscardRows() - 1];
          addr = node_id_t(resource.work_r.idx.layer() - params.GetNumTreeRDiscardRows() - 1,
                           resource.work_r.idx.node() * batch_size / TREE_ARITY +
                           layer_offset * resource_num);
          offset_r = tree_r_address.address(addr);
          for (size_t i = 0; i < C::PARALLEL_SECTORS; i++) {
            to_disk_r->dst[i] = (fr_t*)&tree_r_files[i][partition][offset_r];
            to_disk_r->src[i] = &to_disk_r->data[i * batch_size / TREE_ARITY];
          }
          to_disk_r->size = batch_size / TREE_ARITY;
          to_disk_r->stride = 1;
          to_disk_r->reverse = false;
        }
        
        // Initiate transfer of data to files
        resources[resource_num]->async_done = false;
        cuda_notify.schedule(resource.stream, [this, resource_num,
                                               to_disk, to_disk_r, write_tree_r]() {
          if (!tree_r_only) {
            this->host_buf_to_disk0.enqueue(to_disk);
          }
          if (write_tree_r) {
            this->host_buf_to_disk0.enqueue(to_disk_r);
          }
          resources[resource_num]->async_done = true;
        });

        resource.state = ResourceState::HASH_WAIT;
        break;
      
      case ResourceState::HASH_WAIT:
        if (resource.async_done.load() == true) {
          if (resource.last) {
            resource.state = ResourceState::DONE;
          } else {
            resource.state = ResourceState::IDLE;
          }
        }
        break;

      default:
        abort();
      }
    }
  }
  //printf("PC2: GPU state machine done, syncing streams\n");
  for (size_t resource_num = 0; resource_num < stream_count; resource_num++) {
    resources[resource_num]->stream.sync();
  }

  terminate = true;

  // Really only need this at the last partition...
  if (tree_r_only) {
    while (disk_writer_done < 1) {}
  } else {
    while (disk_writer_done < 2) {}
  }
}

template<class C>
void pc2_t<C>::hash_cpu(fr_t* roots, size_t partition, fr_t* input,
                        std::vector<mmap_t<uint8_t>>* tree_files,
                        size_t file_offset) {
  // This count is one layer above the leaves
  const size_t nodes_to_hash = batch_size * stream_count / TREE_ARITY / TREE_ARITY;
  // Number of consecutive nodes in the input stream
  const size_t group_size = batch_size / TREE_ARITY;
  // For simplicity of indexing require batch size to be a multiple of arity
  assert (group_size % TREE_ARITY == 0);

  tree_address_t final_tree(nodes_to_hash, TREE_ARITY, sizeof(fr_t), 0);

  Poseidon hasher(TREE_ARITY);

  auto hash_func = [this, &hasher, &final_tree, input, partition, tree_files, file_offset, group_size]
    (work_item_t<host_buffer_t>& w) {
    node_id_t addr(w.idx.layer() - 1, w.idx.node());
    size_t offset = final_tree.address(addr) + file_offset;

    if (w.is_leaf) {
      for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
        fr_t* out = &(*w.buf)[sector];
        fr_t in[TREE_ARITY];

        size_t first_input_node = w.idx.node() * TREE_ARITY;
        for (size_t i = 0; i < TREE_ARITY; i++) {
          size_t input_group   = (first_input_node + i) / group_size;
          size_t node_in_group = (first_input_node + i) % group_size;
          
          in[i] = input[input_group * group_size * C::PARALLEL_SECTORS +
                        sector * group_size + node_in_group];
        }
        hasher.Hash((uint8_t*)out, (uint8_t*)in);
        memcpy(&tree_files[sector][partition][offset],
               &out[0], sizeof(fr_t));
      }
    } else {
      for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
        fr_t* out = &(*w.buf)[sector];
        fr_t in[TREE_ARITY];
        for (size_t i = 0; i < TREE_ARITY; i++) {
          in[i] = (*w.inputs[i])[sector];
        }
        hasher.Hash((uint8_t*)out, (uint8_t*)in);
        
        memcpy(&tree_files[sector][partition][offset],
               &out[0], sizeof(fr_t));
      }
    }
  };
  
  buffers_t<host_buffer_t> buffers(C::PARALLEL_SECTORS);
  scheduler_t<host_buffer_t> scheduler(nodes_to_hash, TREE_ARITY, buffers);
  host_buffer_t* host_buf = scheduler.run(hash_func);
  memcpy(roots, &(*host_buf)[0], sizeof(fr_t) * C::PARALLEL_SECTORS);
  assert (scheduler.is_done());
}

template<class C>
void pc2_t<C>::write_roots(fr_t* roots_c, fr_t* roots_r) {
  if (params.GetNumTreeRCFiles() > 1) {
    Poseidon hasher(TREE_ARITY);
    for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
      fr_t in[TREE_ARITY];
      fr_t out_c;
      if (!tree_r_only) {
        for (size_t i = 0; i < TREE_ARITY; i++) {
          in[i] = roots_c[i * C::PARALLEL_SECTORS + sector];
        }
        hasher.Hash((uint8_t*)&out_c, (uint8_t*)in);
      }
      
      fr_t out_r;
      for (size_t i = 0; i < TREE_ARITY; i++) {
        in[i] = roots_r[i * C::PARALLEL_SECTORS + sector];
      }
      hasher.Hash((uint8_t*)&out_r, (uint8_t*)in);
    
      const size_t MAX = 256;
      char fname[MAX];
      if (C::PARALLEL_SECTORS == 1) {
        snprintf(fname, MAX, p_aux_template, output_dir);
      } else {
        snprintf(fname, MAX, p_aux_template, output_dir, sector);
      }
      int p_aux = open(fname, O_RDWR | O_CREAT, (mode_t)0664);
      assert (p_aux != -1);
      if (tree_r_only) {
        fr_t zero;
        zero.zero();
        assert (write(p_aux, &zero, sizeof(fr_t)) == sizeof(fr_t));
      } else {
        assert (write(p_aux, &out_c, sizeof(fr_t)) == sizeof(fr_t));
      }
      assert (write(p_aux, &out_r, sizeof(fr_t)) == sizeof(fr_t));
      close(p_aux);
    }
  } else {
    for (size_t sector = 0; sector < C::PARALLEL_SECTORS; sector++) {
      fr_t out_c = roots_c[sector];
      fr_t out_r = roots_r[sector];

      const size_t MAX = 256;
      char fname[MAX];
      if (C::PARALLEL_SECTORS == 1) {
        snprintf(fname, MAX, p_aux_template, output_dir);
      } else {
        snprintf(fname, MAX, p_aux_template, output_dir, sector);
      }
      int p_aux = open(fname, O_RDWR | O_CREAT, (mode_t)0664);
      assert (p_aux != -1);
      if (tree_r_only) {
        fr_t zero;
        zero.zero();
        assert (write(p_aux, &zero, sizeof(fr_t)) == sizeof(fr_t));
      } else {
        assert (write(p_aux, &out_c, sizeof(fr_t)) == sizeof(fr_t));
      }
      assert (write(p_aux, &out_r, sizeof(fr_t)) == sizeof(fr_t));
      close(p_aux);
    }
  }
}

template<class C>
void pc2_hash(SectorParameters& params, topology_t& topology,
              bool tree_r_only,
              streaming_node_reader_t<C>& reader,
              size_t nodes_to_read, size_t batch_size,
              size_t stream_count,
              const char** data_filenames, const char* output_dir) {
  pc2_t<C> pc2(params, topology, tree_r_only, reader, nodes_to_read, batch_size, stream_count,
               data_filenames, output_dir);
  pc2.hash();
}

template void pc2_hash<sealing_config128_t>(SectorParameters& params, topology_t& topology,
                                            bool tree_r_only,
                                            streaming_node_reader_t<sealing_config128_t>& reader,
                                            size_t nodes_to_read, size_t batch_size,
                                            size_t stream_count,
                                            const char** data_filenames, const char* output_dir);
template void pc2_hash<sealing_config64_t>(SectorParameters& params, topology_t& topology,
                                           bool tree_r_only,
                                           streaming_node_reader_t<sealing_config64_t>& reader,
                                           size_t nodes_to_read, size_t batch_size,
                                           size_t stream_count,
                                           const char** data_filenames, const char* output_dir);
template void pc2_hash<sealing_config32_t>(SectorParameters& params, topology_t& topology,
                                           bool tree_r_only,
                                           streaming_node_reader_t<sealing_config32_t>& reader,
                                           size_t nodes_to_read, size_t batch_size,
                                           size_t stream_count,
                                           const char** data_filenames, const char* output_dir);
template void pc2_hash<sealing_config16_t>(SectorParameters& params, topology_t& topology,
                                           bool tree_r_only,
                                           streaming_node_reader_t<sealing_config16_t>& reader,
                                           size_t nodes_to_read, size_t batch_size,
                                           size_t stream_count,
                                           const char** data_filenames, const char* output_dir);
template void pc2_hash<sealing_config8_t>(SectorParameters& params, topology_t& topology,
                                          bool tree_r_only,
                                          streaming_node_reader_t<sealing_config8_t>& reader,
                                          size_t nodes_to_read, size_t batch_size,
                                          size_t stream_count,
                                          const char** data_filenames, const char* output_dir);
template void pc2_hash<sealing_config4_t>(SectorParameters& params, topology_t& topology,
                                          bool tree_r_only,
                                          streaming_node_reader_t<sealing_config4_t>& reader,
                                          size_t nodes_to_read, size_t batch_size,
                                          size_t stream_count,
                                          const char** data_filenames, const char* output_dir);
template void pc2_hash<sealing_config2_t>(SectorParameters& params, topology_t& topology,
                                          bool tree_r_only,
                                          streaming_node_reader_t<sealing_config2_t>& reader,
                                          size_t nodes_to_read, size_t batch_size,
                                          size_t stream_count,
                                          const char** data_filenames, const char* output_dir);
template void pc2_hash<sealing_config1_t>(SectorParameters& params, topology_t& topology,
                                          bool tree_r_only,
                                          streaming_node_reader_t<sealing_config1_t>& reader,
                                          size_t nodes_to_read, size_t batch_size,
                                          size_t stream_count,
                                          const char** data_filenames, const char* output_dir);

#endif
